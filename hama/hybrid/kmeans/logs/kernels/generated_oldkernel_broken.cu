#define ARRAY_CHECKS

#ifndef NAN

#include <math_constants.h>

#define NAN CUDART_NAN

#endif

#ifndef INFINITY

#include <math_constants.h>

#define INFINITY CUDART_INF

#endif


#include <hip/hip_runtime.h>
#include <stdio.h>

__shared__ size_t m_Local[3];

__shared__ char m_shared[40960];

__device__
int getThreadId(){
  return blockIdx.x * blockDim.x + threadIdx.x;
}
__device__
int getThreadIdxx(){
  return threadIdx.x;
}
__device__
int getBlockIdxx(){
  return blockIdx.x;
}
__device__
int getBlockDimx(){
  return blockDim.x;
}
__device__
int getGridDimx(){
  return blockDim.x;
}
__device__
void edu_syr_pcpratts_syncthreads(){
  __syncthreads();
}
__device__
void edu_syr_pcpratts_threadfence(){
  __threadfence();
}
__device__ clock_t global_now;

/*HAMA_PIPES_HEADER_CODE_IGNORE_IN_TWEAKS_START*/

/* before HostDeviceInterface
nvcc generated.cu --ptxas-options=-v
ptxas info    : 8 bytes gmem, 4 bytes cmem[14]
ptxas info    : Compiling entry function '_Z5entryPcS_PiPxS1_S0_S0_i' for 'sm_10'
ptxas info    : Used 5 registers, 104 bytes smem, 20 bytes cmem[1]

after HostDeviceInterface

nvcc generated.cu --ptxas-options=-v

ptxas info    : 72 bytes gmem, 36 bytes cmem[14]
ptxas info    : Compiling entry function '_Z5entryPcS_PiPxS1_S0_S0_iS0_' for 'sm_10'
ptxas info    : Used 5 registers, 112 bytes smem, 20 bytes cmem[1]


nvcc generated.cu --ptxas-options=-v -arch sm_20

ptxas info    : 72 bytes gmem, 72 bytes cmem[14]
ptxas info    : Compiling entry function '_Z5entryPcS_PiPxS1_S0_S0_iS0_' for 'sm_20'
ptxas info    : Function properties for _Z5entryPcS_PiPxS1_S0_S0_iS0_
    0 bytes stack frame, 0 bytes spill stores, 0 bytes spill loads
ptxas info    : Used 12 registers, 24 bytes smem, 104 bytes cmem[0]

*/

#include <string>

#define STR_SIZE 1024

using std::string;

class HostDeviceInterface {
public:
  volatile bool is_debugging; 

  // Only one thread is able to use the
  // HostDeviceInterface
  volatile int lock_thread_id; 

  // HostMonitor has_task
  volatile bool has_task;

  // HostMonitor is done (end of communication)
  volatile bool done;

  // Request for HostMonitor
  enum MESSAGE_TYPE {
    START_MESSAGE, SET_BSPJOB_CONF, SET_INPUT_TYPES,
    RUN_SETUP, RUN_BSP, RUN_CLEANUP,
    READ_KEYVALUE, WRITE_KEYVALUE,
    GET_MSG, GET_MSG_COUNT,
    SEND_MSG, SYNC,
    GET_ALL_PEERNAME, GET_PEERNAME,
    GET_PEER_INDEX, GET_PEER_COUNT, GET_SUPERSTEP_COUNT,
    REOPEN_INPUT, CLEAR,
    CLOSE, ABORT,
    DONE, TASK_DONE,
    REGISTER_COUNTER, INCREMENT_COUNTER,
    SEQFILE_OPEN, SEQFILE_READNEXT,
    SEQFILE_APPEND, SEQFILE_CLOSE,
    PARTITION_REQUEST, PARTITION_RESPONSE,
    LOG, END_OF_DATA,
    UNDEFINED
  };
  volatile MESSAGE_TYPE command;

  // Command parameter
  volatile bool use_int_val1; // in int_val1
  volatile bool use_int_val2; // in int_val2
  volatile bool use_int_val3; // in int_val3
  volatile bool use_long_val1; // in long_val1
  volatile bool use_long_val2; // in long_val2
  volatile bool use_float_val1; // in float_val1
  volatile bool use_float_val2; // in float_val2
  volatile bool use_double_val1; // in double_val1
  volatile bool use_double_val2; // in double_val2
  volatile bool use_str_val1; // in str_val1
  volatile bool use_str_val2; // in str_val2
  volatile bool use_str_val3; // in str_val3

  // Transfer variables (used in sendCommand and getResult)
  volatile int int_val1;
  volatile int int_val2;
  volatile int int_val3;
  volatile long long_val1;
  volatile long long_val2;
  volatile float float_val1;
  volatile float float_val2;
  volatile double double_val1;
  volatile double double_val2;
  volatile char str_val1[STR_SIZE];
  volatile char str_val2[STR_SIZE];
  volatile char str_val3[255];

  enum TYPE {
    INT, LONG, FLOAT, DOUBLE, STRING, STRING_ARRAY,
    KEY_VALUE_PAIR, NULL_TYPE, NOT_AVAILABLE
  };
  volatile TYPE return_type;
  volatile TYPE key_type;
  volatile TYPE value_type;

  volatile bool end_of_data;

  // Response of HostMonitor
  volatile bool is_result_available;

  HostDeviceInterface() {
    init();
  }

  void init() {
    is_debugging = false;
    lock_thread_id = -1;
    has_task = false;
    done = false;
    command = UNDEFINED;
    use_int_val1 = false;
    use_int_val2 = false;
    use_int_val3 = false;
    use_long_val1 = false;
    use_long_val2 = false;
    use_float_val1 = false;
    use_float_val2 = false;
    use_double_val1 = false;
    use_double_val2 = false;
    use_str_val1 = false;
    use_str_val2 = false;
    use_str_val3 = false;
    int_val1 = 0;
    int_val2 = 0;
    int_val3 = 0;
    long_val1 = 0;
    long_val2 = 0;
    float_val1 = 0;
    float_val2 = 0;
    double_val1 = 0;
    double_val2 = 0;
    key_type = NOT_AVAILABLE;
    value_type = NOT_AVAILABLE;
    end_of_data = true;
    is_result_available = false;
  }

  ~HostDeviceInterface() {}
};

__device__ HostDeviceInterface *host_device_interface;

/*HAMA_PIPES_HEADER_CODE_IGNORE_IN_TWEAKS_END*/
__device__ double java_lang_Math_abs8_8_( char * gc_info, double parameter0, int * exception);

__device__ int java_lang_Float_toString9_7_( char * gc_info, float parameter0, int * exception);

__device__ int java_lang_Object_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void java_lang_Object_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_assignCenters0_5_a12_( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_StringBuilder_append10_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ double java_lang_Math_sqrt8_8_( char * gc_info, double parameter0, int * exception);

__device__ double double__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void double__array_set( char * gc_info, int thisref, int parameter0, double parameter1, int * exception);

__device__ int double__array_new( char * gc_info, int size, int * exception);

__device__ int double__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int int__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void int__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int int__array_new( char * gc_info, int size, int * exception);

__device__ int int__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a660_13_13_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ void edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a66_body0_13_13_( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_AbstractStringBuilder_initab850b60f96d11de8a390800200c9a660_5_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_AbstractStringBuilder_initab850b60f96d11de8a390800200c9a66_body0_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_Integer_getChars0_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_Long_getChars0_6_5_a14_( char * gc_info, long long parameter0, int parameter1, int parameter2, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new( char * gc_info, int size, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int java_lang_Double_toString9_8_( char * gc_info, double parameter0, int * exception);

__device__ int java_lang_Integer_toString9_5_( char * gc_info, int parameter0, int * exception);

__device__ int java_lang_String_length5_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Exception_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_Exception_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_Object_toString9_( char * gc_info, int thisref, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_getLength5_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_Object_toString9_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_add0_11_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_AbstractStringBuilder_append15_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_AbstractStringBuilder_append15_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_StringBuilder_append15_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_DigitOnes( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_DigitOnes( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_DigitTens( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_DigitTens( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_java_lang_String_hash( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_hash( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Class_name( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Class_name( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_AbstractStringBuilder_sizeTable( char * gc_info, int * exception);

__device__ void static_setter_java_lang_AbstractStringBuilder_sizeTable( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount( char * gc_info, int thisref, int parameter0, int * exception);

__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_superstepCount( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_superstepCount( char * gc_info, int thisref, long long parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_digits( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_digits( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int static_getter_java_lang_System_out( char * gc_info, int * exception);

__device__ void static_setter_java_lang_System_out( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int parameter0, int * exception);

__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged( char * gc_info, int thisref, long long parameter0, int * exception);

__device__ int static_getter_java_lang_Integer_sizeTable( char * gc_info, int * exception);

__device__ void static_setter_java_lang_Integer_sizeTable( char * gc_info, int parameter0, int * expcetion);

__device__ int instance_getter_java_lang_String_count( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_count( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_String_value( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_value( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_maxIterations( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_maxIterations( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_String_offset( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_String_offset( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Throwable_cause( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Throwable_cause( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtime_Sentinal_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void edu_syr_pcpratts_rootbeer_runtime_Sentinal_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_String__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_String__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_String__array_new( char * gc_info, int size, int * exception);

__device__ int java_lang_String__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a66( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_String_initab850b60f96d11de8a390800200c9a66_body( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Object_hashCode( char * gc_info, int thisref, int * exception);

__device__ int java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_getKey13_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_AbstractStringBuilder_append15_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception);

__device__ int invoke_java_lang_Object_hashCode( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Long_toString9_6_( char * gc_info, long long parameter0, int * exception);

__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_String_initab850b60f96d11de8a390800200c9a66_body0_5_5_a14_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception);

__device__ void java_lang_String_getChars0_5_5_a14_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int parameter3, int * exception);

__device__ int java_lang_Throwable_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_Throwable_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Object_toString9_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_String_initab850b60f96d11de8a390800200c9a66_body0_a14_5_5_( char * gc_info, int thisref, int parameter0, int parameter1, int parameter2, int * exception);

__device__ void java_lang_AbstractStringBuilder_expandCapacity0_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_AbstractStringBuilder_append15_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int invoke_java_lang_StringBuilder_append15_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ double at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_measureEuclidianDistance8_a12_a12_( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_Integer_stringSize5_5_( char * gc_info, int parameter0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int invoke_java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Math_min5_5_5_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ char char__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void char__array_set( char * gc_info, int thisref, int parameter0, char parameter1, int * exception);

__device__ int char__array_new( char * gc_info, int size, int * exception);

__device__ int char__array_new_multi_array( char * gc_info, int dim0, int * exception);

__device__ int java_lang_Integer_toHexString9_5_( char * gc_info, int parameter0, int * exception);

__device__ int java_lang_Integer_toUnsignedString9_5_5_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_gpuMethod0_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_Long_stringSize5_6_( char * gc_info, long long parameter0, int * exception);

__device__ int java_util_Arrays_copyOfRangea14_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception);

__device__ int java_lang_Boolean_toString9_1_( char * gc_info, char parameter0, int * exception);

__device__ void 
java_lang_System_arraycopy( char * gc_info, int src_handle, int srcPos, int dest_handle, int destPos, int length, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_getLength5_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_StringBuilder_append10_9_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int java_util_Arrays_copyOfa14_a14_5_( char * gc_info, int parameter0, int parameter1, int * exception);

__device__ int double__array__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void double__array__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int double__array__array_new( char * gc_info, int size, int * exception);

__device__ int double__array__array_new_multi_array( char * gc_info, int dim0, int dim1, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_getNearestCenter5_a12_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Character_toString9_3_( char * gc_info, char parameter0, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception);

__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_get11_5_( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_AbstractStringBuilder_stringSizeOfInt5_5_( char * gc_info, int parameter0, int * exception);

__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_( char * gc_info, int thisref, int * exception);

__device__ int java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_( char * gc_info, int parameter0, int * exception);

__device__ void java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_5_( char * gc_info, int thisref, int parameter0, int * exception);

#define GC_OBJ_TYPE_COUNT char

#define GC_OBJ_TYPE_COLOR char

#define GC_OBJ_TYPE_TYPE int

#define GC_OBJ_TYPE_CTOR_USED char

#define GC_OBJ_TYPE_SIZE int

#define COLOR_GREY 0

#define COLOR_BLACK 1

#define COLOR_WHITE 2

__device__ void edu_syr_pcpratts_gc_collect( char * gc_info);

__device__ void edu_syr_pcpratts_gc_assign( char * gc_info, int * lhs, int rhs);

__device__  char * edu_syr_pcpratts_gc_deref( char * gc_info, int handle);

__device__ int edu_syr_pcpratts_gc_malloc( char * gc_info, int size);

__device__ unsigned long long edu_syr_pcpratts_gc_malloc_no_fail( char * gc_info, int size);

__device__ int edu_syr_pcpratts_classConstant(int type_num);

__device__ long long java_lang_System_nanoTime( char * gc_info, int * exception);

#define CACHE_SIZE_BYTES 32

#define CACHE_SIZE_INTS (CACHE_SIZE_BYTES / sizeof(int))

#define CACHE_ENTRY_SIZE 4

#define TO_SPACE_OFFSET               0

#define TO_SPACE_FREE_POINTER_OFFSET  8

__device__
void edu_syr_pcpratts_exitMonitorMem( char * gc_info, char * mem, int old){
  if(old == -1){
   
    edu_syr_pcpratts_threadfence(); 
    atomicExch((int *) mem, -1);
  }
}
__device__ double java_lang_StrictMath_sqrt( char * gc_info , double parameter0 , int * exception ) {
 
  return sqrt(parameter0); 
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx( char * gc_info, int * exception){
  return getThreadIdxx();
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getBlockIdxx( char * gc_info, int * exception){
  return getBlockIdxx();
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getBlockDimx( char * gc_info, int * exception){
  return getBlockDimx();
}
__device__ 
int edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getGridDimx( char * gc_info, int * exception){
  return getGridDimx();
}
__device__
void java_io_PrintStream_println0_9_( char * gc_info, int thisref, int str_ret, int * exception){
  int valueref;
  int count;
  int offset;
  int i;
  int curr_offset;
  char * valueref_deref;
  valueref = instance_getter_java_lang_String_value(gc_info, str_ret, exception);  
  if(*exception != 0){
    return; 
  }
 
  count = instance_getter_java_lang_String_count(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  offset = instance_getter_java_lang_String_offset(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  valueref_deref = (char *) edu_syr_pcpratts_gc_deref(gc_info, valueref);
  for(i = offset; i < count; ++i){
    curr_offset = 32 + (i * 4);
    printf("%c", valueref_deref[curr_offset]);
  }
  printf("\n");
}
__device__
void java_io_PrintStream_println0_6_( char * gc_info, int thisref, long long value, int * exception){
  printf("%lld\n", value);
}
__device__
void java_io_PrintStream_println0_8_( char * gc_info, int thisref, double value, int * exception){
  printf("%e\n", value);
}
__device__
void java_io_PrintStream_print0_9_( char * gc_info, int thisref, int str_ret, int * exception){
  int valueref;
  int count;
  int offset;
  int i;
  int curr_offset;
  char * valueref_deref;
  valueref = instance_getter_java_lang_String_value(gc_info, str_ret, exception);  
  if(*exception != 0){
    return; 
  }
 
  count = instance_getter_java_lang_String_count(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  offset = instance_getter_java_lang_String_offset(gc_info, str_ret, exception);
  if(*exception != 0){
    return; 
  }
 
  valueref_deref = (char *) edu_syr_pcpratts_gc_deref(gc_info, valueref);
  for(i = offset; i < count; ++i){
    curr_offset = 32 + (i * 4);
    printf("%c", valueref_deref[curr_offset]);
  }
}
__device__ 
void edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads( char * gc_info, int * exception){
  edu_syr_pcpratts_syncthreads();
}
__device__ char
edu_syr_pcpratts_cmp(long long lhs, long long rhs){
  if(lhs > rhs)
    return 1;
  if(lhs < rhs)
    return -1;
  return 0;
}
__device__ char
edu_syr_pcpratts_cmpl(double lhs, double rhs){
  if(lhs > rhs)
    return 1;
  if(lhs < rhs)
    return -1;
  if(lhs == rhs)
    return 0;
  return -1;
}
__device__ char
edu_syr_pcpratts_cmpg(double lhs, double rhs){
  if(lhs > rhs)
    return 1;
  if(lhs < rhs)
    return -1;
  if(lhs == rhs)
    return 0;
  return 1;
}
__device__ void
edu_syr_pcpratts_gc_set_count( char * mem_loc, GC_OBJ_TYPE_COUNT value){
  mem_loc[0] = value;
}
__device__ void
edu_syr_pcpratts_gc_set_color( char * mem_loc, GC_OBJ_TYPE_COLOR value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT);
  mem_loc[0] = value;
}
__device__ void
edu_syr_pcpratts_gc_init_monitor( char * mem_loc){
  int * addr;
  mem_loc += 16;
  addr = (int *) mem_loc;
  *addr = -1;
}
__device__ void
edu_syr_pcpratts_gc_set_type( char * mem_loc, GC_OBJ_TYPE_TYPE value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char) +
    sizeof(GC_OBJ_TYPE_CTOR_USED);
  *(( GC_OBJ_TYPE_TYPE *) &mem_loc[0]) = value;
}
__device__ GC_OBJ_TYPE_TYPE
edu_syr_pcpratts_gc_get_type( char * mem_loc){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char) +
    sizeof(GC_OBJ_TYPE_CTOR_USED);
  return *(( GC_OBJ_TYPE_TYPE *) &mem_loc[0]);
}
__device__ void
edu_syr_pcpratts_gc_set_ctor_used( char * mem_loc, GC_OBJ_TYPE_CTOR_USED value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char);
  mem_loc[0] = value;
}
__device__ void
edu_syr_pcpratts_gc_set_size( char * mem_loc, GC_OBJ_TYPE_SIZE value){
  mem_loc += sizeof(GC_OBJ_TYPE_COUNT) + sizeof(GC_OBJ_TYPE_COLOR) + sizeof(char) + 
    sizeof(GC_OBJ_TYPE_CTOR_USED) + sizeof(GC_OBJ_TYPE_TYPE);
  *(( GC_OBJ_TYPE_SIZE *) &mem_loc[0]) = value;
}
__device__ int edu_syr_pcpratts_getint( char * buffer, int pos){
  return *(( int *) &buffer[pos]);
}
__device__ void edu_syr_pcpratts_setint( char * buffer, int pos, int value){
  *(( int *) &buffer[pos]) = value;
}
__device__ int
edu_syr_pcpratts_strlen(char * str_constant){
  int ret = 0;
  while(1){
    if(str_constant[ret] != 
'\0'
){
      ret++;
    }
 else {
      return ret;
    }
  }
}
__device__ int
edu_syr_pcpratts_array_length( char * gc_info, int thisref){
  
  
  
  
  
     char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
    int ret = edu_syr_pcpratts_getint(thisref_deref, 12);
    return ret;
  
}
__device__
int java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(char * gc_info, int * exception){
 
  int thisref;
  char * thisref_deref;
  int chars;
  thisref = edu_syr_pcpratts_gc_malloc(gc_info , 48);
  if(thisref == -1){
    *exception = 21352; 
    return -1; 
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  edu_syr_pcpratts_gc_set_count(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY); 
  edu_syr_pcpratts_gc_set_type(thisref_deref, 2905); 
  edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_size(thisref_deref, 48); 
  edu_syr_pcpratts_gc_init_monitor(thisref_deref); 
  chars = char__array_new(gc_info, 0, exception);
  instance_setter_java_lang_AbstractStringBuilder_value(gc_info, thisref, chars, exception); 
  instance_setter_java_lang_AbstractStringBuilder_count(gc_info, thisref, 0, exception);
  return thisref; 
}
__device__
int java_lang_String_initab850b60f96d11de8a390800200c9a66(char * gc_info, int parameter0, int * exception) {
 
  int r0 = -1; 
  int r1 = -1; 
  int i0; 
  int $r2 = -1; 
  int thisref; 
  char * thisref_deref; 
  int i;
  int len;
  int characters_copy;
  char ch;
  
  thisref = -1; 
  edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48)); 
  if(thisref == -1) {
 
    *exception = 21352; 
    return -1; 
  }
 
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref); 
  edu_syr_pcpratts_gc_set_count(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY); 
  edu_syr_pcpratts_gc_set_type(thisref_deref, 2905); 
  edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1); 
  edu_syr_pcpratts_gc_set_size(thisref_deref, 48); 
  edu_syr_pcpratts_gc_init_monitor(thisref_deref); 
  len = edu_syr_pcpratts_array_length(gc_info, parameter0);
  characters_copy = char__array_new(gc_info, len, exception);
  for(i = 0; i < len; ++i){
    ch = char__array_get(gc_info, parameter0, i, exception);
    char__array_set(gc_info, characters_copy, i, ch, exception);
  }
  instance_setter_java_lang_String_value(gc_info, thisref, characters_copy, exception); 
  instance_setter_java_lang_String_count(gc_info, thisref, len, exception); 
  instance_setter_java_lang_String_offset(gc_info, thisref, 0, exception); 
  return thisref; 
}
__device__ int 
char__array_new( char * gc_info, int size, int * exception);

__device__ void 
char__array_set( char * gc_info, int thisref, int parameter0, char parameter1, int * exception);

__device__ int
edu_syr_pcpratts_string_constant( char * gc_info, char * str_constant, int * exception){
  int i;
  int len = edu_syr_pcpratts_strlen(str_constant);
  int characters = char__array_new(gc_info, len, exception);
  unsigned long long * addr = (unsigned long long *) (gc_info + TO_SPACE_FREE_POINTER_OFFSET);
  for(i = 0; i < len; ++i){
    char__array_set(gc_info, characters, i, str_constant[i], exception);
  }
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, characters, exception);
}
__device__ void
edu_syr_pcpratts_gc_assign( char * gc_info, int * lhs_ptr, int rhs){
  *lhs_ptr = rhs;
}
__device__ int java_lang_StackTraceElement__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_StackTraceElement__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception);

__device__ int java_lang_StackTraceElement__array_new( char * gc_info, int size, int * exception);

__device__ int java_lang_StackTraceElement_initab850b60f96d11de8a390800200c9a660_3_3_3_4_( char * gc_info, int parameter0, int parameter1, int parameter2, int parameter3, int * exception);

__device__ void instance_setter_java_lang_RuntimeException_stackDepth( char * gc_info, int thisref, int parameter0);

__device__ int instance_getter_java_lang_RuntimeException_stackDepth( char * gc_info, int thisref);

__device__ int java_lang_StackTraceElement__array_get( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int instance_getter_java_lang_Throwable_stackTrace( char * gc_info, int thisref, int * exception);

__device__ void instance_setter_java_lang_Throwable_stackTrace( char * gc_info, int thisref, int parameter0, int * exception);

__device__ int java_lang_Throwable_fillInStackTrace( char * gc_info, int thisref, int * exception){
  
  
  return thisref;
}
__device__ void instance_setter_java_lang_Throwable_cause( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void instance_setter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void instance_setter_java_lang_Throwable_stackDepth( char * gc_info, int thisref, int parameter0, int * exception);

__device__ void java_lang_VirtualMachineError_initab850b60f96d11de8a390800200c9a66_body0_( char * gc_info, int thisref, int * exception);

__device__ int
java_lang_Object_hashCode( char * gc_info, int thisref, int * exception){
  return thisref;
}
__device__ int
java_lang_Class_getName( char * gc_info , int thisref , int * exception ) {
 
  int $r1 =-1 ; 
  $r1 = instance_getter_java_lang_Class_name ( gc_info , thisref , exception ) ; 
  if ( * exception != 0 ) {
 
    return 0 ; 
  }
 
  return $r1;
}
__device__ int
java_lang_Object_getClass( char * gc_info , int thisref, int * exception ) {
 
  char * mem_loc = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  int type = edu_syr_pcpratts_gc_get_type(mem_loc);
  return edu_syr_pcpratts_classConstant(type);
}
__device__ 
int java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a6610_9_(char * gc_info, 
  int str ,int * exception){
 
  int r0 = -1; 
  int thisref; 
  int str_value;
  int str_count;  
  char * thisref_deref; 
  thisref = -1;
  edu_syr_pcpratts_gc_assign ( gc_info , & thisref , edu_syr_pcpratts_gc_malloc ( gc_info , 48 ) ) ; 
  if ( thisref ==-1 ) {
 
    * exception = 21352; 
    return-1 ; 
  }
 
  thisref_deref = edu_syr_pcpratts_gc_deref ( gc_info , thisref ) ; 
  edu_syr_pcpratts_gc_set_count ( thisref_deref , 0 ) ; 
  edu_syr_pcpratts_gc_set_color ( thisref_deref , COLOR_GREY ) ; 
  edu_syr_pcpratts_gc_set_type ( thisref_deref , 16901 ) ; 
  edu_syr_pcpratts_gc_set_ctor_used ( thisref_deref , 1 ) ; 
  edu_syr_pcpratts_gc_set_size ( thisref_deref , 44 ) ; 
  edu_syr_pcpratts_gc_init_monitor ( thisref_deref ) ; 
  str_value = instance_getter_java_lang_String_value(gc_info, str, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str, exception);
  instance_setter_java_lang_AbstractStringBuilder_value(gc_info, thisref, str_value, exception); 
  instance_setter_java_lang_AbstractStringBuilder_count(gc_info, thisref, str_count, exception); 
  return thisref; 
}
__device__ 
int java_lang_StringBuilder_append10_9_(char * gc_info, int thisref,
  int parameter0, int * exception){
  int sb_value;
  int sb_count;
  int str_value;
  int str_count;
  int new_count;
  int new_sb_value;
  int i;
  char ch;
  int new_str;
  
  sb_value = instance_getter_java_lang_AbstractStringBuilder_value(gc_info, thisref,
    exception);
  sb_count = instance_getter_java_lang_AbstractStringBuilder_count(gc_info, thisref,
    exception);
  
  str_value = instance_getter_java_lang_String_value(gc_info, parameter0,
    exception);
  str_count = instance_getter_java_lang_String_count(gc_info, parameter0,
    exception);
  new_count = sb_count + str_count;
  new_sb_value = char__array_new(gc_info, new_count, exception);
  for(i = 0; i < sb_count; ++i){
    ch = char__array_get(gc_info, sb_value, i, exception);
    char__array_set(gc_info, new_sb_value, i, ch, exception);
  }
  for(i = 0; i < str_count; ++i){
    ch = char__array_get(gc_info, str_value, i, exception);
    char__array_set(gc_info, new_sb_value, sb_count + i, ch, exception);
  }
  
  new_str = java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, 
    new_sb_value, exception);
  
  return java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a6610_9_(gc_info,
    new_str, exception);
}
__device__ 
int java_lang_StringBuilder_append10_5_(char * gc_info, int thisref,
  int parameter0, int * exception){
  int str = java_lang_Integer_toString9_5_(gc_info, parameter0, exception);
  return java_lang_StringBuilder_append10_9_(gc_info, thisref, str, exception);
}
__device__ 
int java_lang_StringBuilder_toString9_(char * gc_info, int thisref,
  int * exception){
 
  int value = instance_getter_java_lang_AbstractStringBuilder_value(gc_info, thisref,
    exception);
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, value, 
    exception);
}
/*****************************************************************************/
/* local methods */
__device__
int at_illecker_strlen(volatile char * str_constant) {
  int ret = 0;
  while(1) {
    if(str_constant[ret] != 
'\0'
) {
      ret++;
    }
 else {
      return ret;
    }
  }
}
__device__
int at_illecker_string_constant(char * gc_info, volatile char * str_constant, int * exception) {
  if (str_constant == 0) {
    return 0;
  }
  int i;
  int len = at_illecker_strlen(str_constant);
  int characters = char__array_new(gc_info, len, exception);
  
  
  
  
  for(i = 0; i < len; ++i) {
    char__array_set(gc_info, characters, i, str_constant[i], exception);
    
    
    
  }
  
  
  
  
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, characters, exception);
}
/*****************************************************************************/
/* toString methods */
__device__
double at_illecker_abs_val(double value) {
  double result = value;
  if (value < 0) {
    result = -value;
  }
  return result;
}
__device__
double at_illecker_pow10(int exp) {
  double result = 1;
  while (exp) {
    result *= 10;
    exp--;
  }
  return result;
}
__device__
long at_illecker_round(double value) {
  long intpart;
  intpart = value;
  value = value - intpart;
  if (value >= 0.5) {
    intpart++;
  }
  return intpart;
}
__device__
void at_illecker_set_char(char *buffer, int *currlen, int maxlen, char c) {
  if (*currlen < maxlen) {
    buffer[(*currlen)++] = c;
  }
}
__device__
int at_illecker_double_to_string(char * gc_info, double fvalue, int max, int * exception) {
  int signvalue = 0;
  double ufvalue;
  long intpart;
  long fracpart;
  char iconvert[20];
  char fconvert[20];
  int iplace = 0;
  int fplace = 0;
  int zpadlen = 0; 
  char buffer[64];
  int maxlen = 64;
  int currlen = 0;
  
  
  
  if (max < 0) {
    max = 6;
  }
  
  
  if (max > 9) {
    max = 9;
  }
  
  if (fvalue < 0) {
    signvalue = 
'-'
;
  }
  ufvalue = at_illecker_abs_val(fvalue);
  intpart = ufvalue;
  
  
  fracpart = at_illecker_round(at_illecker_pow10(max) * (ufvalue - intpart));
  if (fracpart >= at_illecker_pow10(max)) {
    intpart++;
    fracpart -= at_illecker_pow10(max);
  }
  
  
  
  do {
    iconvert[iplace++] = "0123456789abcdef"[intpart % 10];
    intpart = (intpart / 10);
  }
 while(intpart && (iplace < 20));
  if (iplace == 20) {
    iplace--;
  }
  iconvert[iplace] = 0;
  
  do {
    fconvert[fplace++] = "0123456789abcdef"[fracpart % 10];
    fracpart = (fracpart / 10);
  }
 while(fracpart && (fplace < 20));
  
  if (fplace == 20) {
    fplace--;
  }
  fconvert[fplace] = 0;
  
  zpadlen = max - fplace;
  if (zpadlen < 0) {
    zpadlen = 0;
  }
  
  
  
  if (signvalue) {
    at_illecker_set_char(buffer, &currlen, maxlen, signvalue);
  }
  
  while (iplace > 0) {
    at_illecker_set_char(buffer, &currlen, maxlen, iconvert[--iplace]);
  }
  
  if (max > 0) {
    
    
    
    at_illecker_set_char(buffer, &currlen, maxlen, 
'.'
);
    while (fplace > 0) {
      at_illecker_set_char(buffer, &currlen, maxlen, fconvert[--fplace]);
    }
  }
  
  while (zpadlen > 0) {
    at_illecker_set_char(buffer, &currlen, maxlen, 
'0'
);
    --zpadlen;
  }
  
  if (currlen < maxlen - 1) {
    buffer[currlen] = 
'\0'
;
  }
 else {
    buffer[maxlen - 1] = 
'\0'
;
  }
  return at_illecker_string_constant(gc_info, buffer, exception);
}
__device__ 
int java_lang_Double_toString9_8_(char * gc_info, double double_val, int * exception) {
  
  return at_illecker_double_to_string(gc_info, double_val, 6, exception);
}
/*****************************************************************************/
/* String.indexOf methods */
__device__
int at_illecker_strpos(char * gc_info, int str_value, int str_count, 
                       int sub_str_value, int sub_str_count, 
                       int start_pos, int * exception) {
  if ( (str_count == 0) || (sub_str_count == 0) || 
       (start_pos > str_count)) {
    return -1;
  }
  for (int i = start_pos; i < str_count; i++) {
    if (char__array_get(gc_info, str_value, i, exception) != 
        char__array_get(gc_info, sub_str_value, 0, exception)) {
      continue;
    }
    int found_pos = i;
    int found_sub_string = true;
    for (int j = 1; j < sub_str_count; j++) {
      i++;
      if (char__array_get(gc_info, str_value, i, exception) != 
          char__array_get(gc_info, sub_str_value, j, exception)) {
        found_sub_string = false;
        break;
      }
    }
    if (found_sub_string) {
      return found_pos;
    }
  }
  return -1;
}
/*****************************************************************************/
/* String.substring methods */
__device__
int at_illecker_substring(char * gc_info, int str_value, int str_count, 
                       int begin_index, int end_index, int * exception) {
  int new_length = 0;
  int new_string = -1;
  
  if (end_index == -1) {
 
    new_length = str_count - begin_index;
  }
 else {
    if (end_index < str_count) {
      new_length = end_index - begin_index;
    }
 else {
      new_length = str_count - begin_index;
    }
  }
 
  
  new_string = char__array_new(gc_info, new_length, exception);
  for(int i = 0; i < new_length; i++) {
    char__array_set(gc_info, new_string, i, char__array_get(gc_info, str_value, begin_index, exception), exception);
    begin_index++;
  }
  return java_lang_String_initab850b60f96d11de8a390800200c9a66(gc_info, new_string, exception);
}
/*****************************************************************************/
/* String.split methods */
__device__
int at_illecker_strcnt(char * gc_info, int str_value, int str_count, 
                       int sub_str_value, int sub_str_count, int * exception) {
  int occurrences = 0;
  if ( (str_count == 0) || (sub_str_count == 0) ) {
    return 0;
  }
  for (int i = 0; i < str_count; i++) {
    if (char__array_get(gc_info, str_value, i, exception) != 
        char__array_get(gc_info, sub_str_value, 0, exception)) {
      continue;
    }
    bool found_sub_string = true;
    for (int j = 1; j < sub_str_count; j++) {
      i++;
      if (char__array_get(gc_info, str_value, i, exception) != 
          char__array_get(gc_info, sub_str_value, j, exception)) {
        found_sub_string = false;
        break;
      }
    }
    if (found_sub_string) {
      occurrences++;
    }
  }
  return occurrences;
}
__device__
int at_illecker_split(char * gc_info, int str_obj_ref, int delim_str_obj_ref,
                      int limit, int * exception) {
  int return_obj = -1;
  int start = 0;
  int end = 0;
  int str_value = 0;
  int str_count = 0;
  int delim_str_value = 0;
  int delim_str_count = 0;
  int delim_occurrences = 0;
  str_value = instance_getter_java_lang_String_value(gc_info, str_obj_ref, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str_obj_ref, exception);
  delim_str_value = instance_getter_java_lang_String_value(gc_info, delim_str_obj_ref, exception);
  delim_str_count = instance_getter_java_lang_String_count(gc_info, delim_str_obj_ref, exception);
  
  delim_occurrences = at_illecker_strcnt(gc_info, str_value, str_count, 
                                         delim_str_value, delim_str_count, exception);
  
  if ( (limit <= 0) || (limit > delim_occurrences) ) {
    return_obj = java_lang_String__array_new(gc_info, delim_occurrences + 1, exception);
    limit = delim_occurrences + 1;
  }
 else {
    return_obj = java_lang_String__array_new(gc_info, limit, exception);
  }
  if (delim_occurrences == 0) {
    
    java_lang_String__array_set(gc_info, return_obj, 0, str_obj_ref, exception);
    
  }
 else {
    
    for (int i = 0; i < limit - 1; i++) {
      end = at_illecker_strpos(gc_info, str_value, str_count, 
                               delim_str_value, delim_str_count, start, exception);
      if (end == -1) {
        break;
      }
      
      java_lang_String__array_set(gc_info, return_obj, i,
        at_illecker_substring(gc_info, str_value, str_count, start, end, exception), exception);
      
      start = end + delim_str_count;
    }
    
    if (end != -1) {
      
      java_lang_String__array_set(gc_info, return_obj, limit - 1,
        at_illecker_substring(gc_info, str_value, str_count, start, -1, exception), exception);
    }
  }
  return return_obj;
}
__device__
int java_lang_String_split(char * gc_info, int str_obj_ref, int delim_str_obj_ref, int limit, int * exception) {
  return at_illecker_split(gc_info, str_obj_ref, delim_str_obj_ref, limit, exception);
}
__device__
int java_lang_String_split(char * gc_info, int str_obj_ref, int delim_str_obj_ref, int * exception) {
  return at_illecker_split(gc_info, str_obj_ref, delim_str_obj_ref, 0, exception);
}
/*****************************************************************************/
/* Parse methods */
__device__
bool at_illecker_is_digit(unsigned char c) {
  return ((c)>=
'0'
 && (c)<=
'9'
);
}
__device__
bool at_illecker_is_space(unsigned char c) {
  return ((c)==
' '
 || (c)==
'\f'
 || (c)==
'\n'
 || (c)==
'\r'
 || (c)==
'\t'
 || (c)==
'\v'
);
}
/* Argument1: String of ASCII digits, possibly
 * preceded by white space.  For bases
 * greater than 10, either lower- or
 * upper-case digits may be used.
 */
/* Argument2: Where to store address of terminating
 * character, or NULL.
 */
/* Argument3: Base for conversion.  Must be less
 * than 37.  If 0, then the base is chosen
 * from the leading characters of string:
 * "0x" means hex, "0" means octal, anything
 * else means decimal.
 */
__device__
unsigned long int at_illecker_strtoul(const char *string, char **end_ptr, int base) {
  register const char *p;
  register unsigned long int result = 0;
  register unsigned digit;
  int anyDigits = 0;
  int negative=0;
  int overflow=0;
  char cvtIn[] = {
    0, 1, 2, 3, 4, 5, 6, 7, 8, 9,		/* 
'0'
 - 
'9'
 */
    100, 100, 100, 100, 100, 100, 100,		/* punctuation */
    10, 11, 12, 13, 14, 15, 16, 17, 18, 19,	/* 
'A'
 - 
'Z'
 */
    20, 21, 22, 23, 24, 25, 26, 27, 28, 29,
    30, 31, 32, 33, 34, 35,
    100, 100, 100, 100, 100, 100,		/* punctuation */
    10, 11, 12, 13, 14, 15, 16, 17, 18, 19,	/* 
'a'
 - 
'z'
 */
    20, 21, 22, 23, 24, 25, 26, 27, 28, 29,
    30, 31, 32, 33, 34, 35
  }
;
  
  p = string;
  while (at_illecker_is_space((unsigned char) (*p))) {
    p += 1;
  }
  
  if (*p == 
'-'
) {
    negative = 1;
    p += 1;
  }
 else {
    if (*p == 
'+'
) {
      p += 1;
    }
  }
  
  
  if (base == 0) {
    if (*p == 
'0'
) {
      p += 1;
      if ((*p == 
'x'
) || (*p == 
'X'
)) {
        p += 1;
        base = 16;
      }
 else {
        
        
        anyDigits = 1;
        base = 8;
      }
    }
 else {
      base = 10;
    }
  }
 else if (base == 16) {
    
    if ((p[0] == 
'0'
) && ((p[1] == 
'x'
) || (p[1] == 
'X'
))) {
      p += 2;
    }
  }
  
  
  if (base == 8) {
    unsigned long maxres = 0xFFFFFFFFUL >> 3; 
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > 7) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result = (result << 3);
      if (digit > (0xFFFFFFFFUL - result)) {
 
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
 else if (base == 10) {
    unsigned long maxres = 0xFFFFFFFFUL / 10; 
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > 9) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result *= 10;
      if (digit > (0xFFFFFFFFUL - result)) {
 
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
 else if (base == 16) {
    unsigned long maxres = 0xFFFFFFFFUL >> 4;
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > (
'z'
 - 
'0'
)) {
        break;
      }
      digit = cvtIn[digit];
      if (digit > 15) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result = (result << 4);
      if (digit > (0xFFFFFFFFUL - result)) {
 
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
 else if ( base >= 2 && base <= 36 ) {
    unsigned long maxres = 0xFFFFFFFFUL / base;
    for ( ; ; p += 1) {
      digit = *p - 
'0'
;
      if (digit > (
'z'
 - 
'0'
)) {
        break;
      }
      digit = cvtIn[digit];
      if (digit >= ( (unsigned) base )) {
        break;
      }
      if (result > maxres) {
 
        overflow = 1;
      }
      result *= base;
      if (digit > (0xFFFFFFFFUL - result)) {
        overflow = 1;
      }
      result += digit;
      anyDigits = 1;
    }
  }
  
  if (!anyDigits) {
    p = string;
  }
  if (end_ptr != 0) {
    /* unsafe, but required by the strtoul prototype */
    *end_ptr = (char *) p;
  }
  if (overflow) {
    
    return 0xFFFFFFFFUL;
  }
 
  if (negative) {
    return -result;
  }
  return result;
}
/* Argument1: String of ASCII digits, possibly
 * preceded by white space.  For bases
 * greater than 10, either lower- or
 * upper-case digits may be used.
 */
/* Argument2: Where to store address of terminating
 * character, or NULL.
 */
/* Argument3: Base for conversion.  Must be less
 * than 37.  If 0, then the base is chosen
 * from the leading characters of string:
 * "0x" means hex, "0" means octal, anything
 * else means decimal.
 */
__device__
long int at_illecker_strtol(const char *string, char **end_ptr, int base) {
  register const char *p;
  long result;
  
  p = string;
  while (at_illecker_is_space((unsigned char) (*p))) {
    p += 1;
  }
  
  if (*p == 
'-'
) {
    p += 1;
    result = -(at_illecker_strtoul(p, end_ptr, base));
  }
 else {
    if (*p == 
'+'
) {
      p += 1;
    }
    result = at_illecker_strtoul(p, end_ptr, base);
  }
  if ((result == 0) && (end_ptr != 0) && (*end_ptr == p)) {
    *end_ptr = (char *) string;
  }
  return result;
}
__device__
double at_illecker_strtod(const char *string) {
  int sign = 0; 
  int expSign = 0; 
  double fraction, dblExp, *d;
  register const char *p;
  register int c;
  int exp = 0;
  int fracExp = 0;
  int mantSize;
  int decPt;
  const char *pExp;
  int maxExponent = 511;
  double powersOf10[] = {
    10.,
    100.,
    1.0e4,
    1.0e8,
    1.0e16,
    1.0e32,
    1.0e64,
    1.0e128,
    1.0e256
  }
;
  
  p = string;
  while (at_illecker_is_space((unsigned char) (*p))) {
    p += 1;
  }
  
  if (*p == 
'-'
) {
    sign = 1; 
    p += 1;
  }
 else {
    if (*p == 
'+'
) {
      p += 1;
    }
    sign = 0; 
  }
  
  
  decPt = -1;
  for (mantSize = 0; ; mantSize += 1) {
    c = *p;
    if (!at_illecker_is_digit(c)) {
      if ((c != 
'.'
) || (decPt >= 0)) {
        break;
      }
      decPt = mantSize;
    }
    p += 1;
  }
  
  
  
  
  pExp  = p;
  p -= mantSize;
  if (decPt < 0) {
    decPt = mantSize;
  }
 else {
    mantSize -= 1;
  }
  if (mantSize > 18) {
    fracExp = decPt - 18;
    mantSize = 18;
  }
 else {
    fracExp = decPt - mantSize;
  }
  if (mantSize == 0) {
    fraction = 0.0;
    p = string;
    goto done;
  }
 else {
    int frac1, frac2;
    frac1 = 0;
    for ( ; mantSize > 9; mantSize -= 1) {
      c = *p;
      p += 1;
      if (c == 
'.'
) {
        c = *p;
        p += 1;
      }
      frac1 = 10*frac1 + (c - 
'0'
);
    }
    frac2 = 0;
    for (; mantSize > 0; mantSize -= 1) {
      c = *p;
      p += 1;
      if (c == 
'.'
) {
        c = *p;
        p += 1;
      }
      frac2 = 10*frac2 + (c - 
'0'
);
    }
    fraction = (1.0e9 * frac1) + frac2;
  }
  
  p = pExp;
  if ((*p == 
'E'
) || (*p == 
'e'
)) {
    p += 1;
    if (*p == 
'-'
) {
      expSign = 1; 
      p += 1;
    }
 else {
      if (*p == 
'+'
) {
        p += 1;
      }
      expSign = 0; 
    }
    if (!at_illecker_is_digit((unsigned char) (*p))) {
      p = pExp;
      goto done;
    }
    while (at_illecker_is_digit((unsigned char) (*p))) {
      exp = exp * 10 + (*p - 
'0'
);
      p += 1;
    }
  }
  if (expSign) {
    exp = fracExp - exp;
  }
 else {
    exp = fracExp + exp;
  }
  
  
  
  
  if (exp < 0) {
    expSign = 1; 
    exp = -exp;
  }
 else {
    expSign = 0; 
  }
  if (exp > maxExponent) {
    exp = maxExponent;
    
    
  }
  dblExp = 1.0;
  for (d = powersOf10; exp != 0; exp >>= 1, d += 1) {
    if (exp & 01) {
      dblExp *= *d;
    }
  }
  if (expSign) {
    fraction /= dblExp;
  }
 else {
    fraction *= dblExp;
  }
done:
  if (sign) {
    return -fraction;
  }
  return fraction;
}
__device__
long java_lang_Long_parseLong(char * gc_info, int str_obj_ref, int * exception) {
  int str_value = 0;
  int str_count = 0;
  char str_val[255];
  long return_val = 0;
  str_value = instance_getter_java_lang_String_value(gc_info, str_obj_ref, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str_obj_ref, exception);
  
  
  for(int i = 0; i < str_count; i++){
    str_val[i] = char__array_get(gc_info, str_value, i, exception);
  }
  str_val[str_count] = 
'\0'
;
  
  return_val = at_illecker_strtol(str_val, 0, 0);
  
  return return_val;
}
__device__
int java_lang_Integer_parseInt(char * gc_info, int str_obj_ref, int * exception) {
  return java_lang_Long_parseLong(gc_info, str_obj_ref, exception);
}
__device__
double java_lang_Double_parseDouble(char * gc_info, int str_obj_ref, int * exception) {
  int str_value = 0;
  int str_count = 0;
  char str_val[255];
  double return_val = 0;
  str_value = instance_getter_java_lang_String_value(gc_info, str_obj_ref, exception);
  str_count = instance_getter_java_lang_String_count(gc_info, str_obj_ref, exception);
  
  
  for(int i = 0; i < str_count; i++){
    str_val[i] = char__array_get(gc_info, str_value, i, exception);
  }
  str_val[str_count] = 
'\0'
;
  
  return_val = at_illecker_strtod(str_val);
  
  return return_val;
}
/*****************************************************************************/
/* local typeof methods */
__device__ bool at_illecker_typeof_Integer(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12647) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_Long(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12639) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_Float(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12643) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_Double(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==12637) {
    return true;
  }
  return false;
}
__device__ bool at_illecker_typeof_String(char * gc_info, int thisref){
  char * thisref_deref;
  GC_OBJ_TYPE_TYPE type;
  if(thisref == -1){
    return false;
  }
  thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
  type = edu_syr_pcpratts_gc_get_type(thisref_deref);
  if(type==2905) {
    return true;
  }
  return false;
}
/*****************************************************************************/
template<class T>
__device__
T at_illecker_getResult( char * gc_info, 
    HostDeviceInterface::MESSAGE_TYPE cmd, 
    HostDeviceInterface::TYPE return_type, bool use_return_value,
    int key_value_pair_ref, HostDeviceInterface::TYPE key_type, HostDeviceInterface::TYPE value_type,
    int int_param1, bool use_int_param1,
    int int_param2, bool use_int_param2,
    int int_param3, bool use_int_param3,
    long long long_param1, bool use_long_param1,
    long long long_param2, bool use_long_param2,
    float float_param1, bool use_float_param1,
    float float_param2, bool use_float_param2,
    double double_param1, bool use_double_param1,
    double double_param2, bool use_double_param2,
    int str_param1, bool use_str_param1,
    int str_param2, bool use_str_param2,
    int str_param3, bool use_str_param3,
    int * exception) {
  T return_value = 0;
  int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int count = 0;
  int timeout = 0;
  bool done = false;
  int str_param1_value = 0;
  int str_param1_count = 0;
  int str_param2_value = 0;
  int str_param2_count = 0;
  int str_param3_value = 0;
  int str_param3_count = 0;
  int key_obj_ref = 0;
  int value_obj_ref = 0;
  char * key_obj_deref;
  char * value_obj_deref;
  
  while (count < 100) {
    
    if (++timeout > 100000) {
      break;
    }
    __syncthreads();
    
    if (done) {
      break;
    }
    
    int old = atomicCAS((int *) &host_device_interface->lock_thread_id, -1, thread_id);
    
    if (old == -1 || old == thread_id) {
      
      
      if (host_device_interface->is_debugging) {
        printf("gpu_Thread %d GOT LOCK lock_thread_id: %d\n", thread_id,
               host_device_interface->lock_thread_id);
      }
      /***********************************************************************/
      
      int inner_timeout = 0;
      while (host_device_interface->has_task) {
        
        if (++inner_timeout > 10000) {
          break;
        }
      }
      /***********************************************************************/
      
      host_device_interface->command = cmd;
      host_device_interface->return_type = return_type;
      
      if (use_int_param1) {
        host_device_interface->use_int_val1 = true;
        host_device_interface->int_val1 = int_param1;
      }
      if (use_int_param2) {
        host_device_interface->use_int_val2 = true;
        host_device_interface->int_val2 = int_param2;
      }
      if (use_int_param3) {
        host_device_interface->use_int_val3 = true;
        host_device_interface->int_val3 = int_param3;
      }
      if (use_long_param1) {
        host_device_interface->use_long_val1 = true;
        host_device_interface->long_val1 = long_param1;
      }
      if (use_long_param2) {
        host_device_interface->use_long_val2 = true;
        host_device_interface->long_val2 = long_param2;
      }
      if (use_float_param1) {
        host_device_interface->use_float_val1 = true;
        host_device_interface->float_val1 = float_param1;
      }
      if (use_float_param2) {
        host_device_interface->use_float_val2 = true;
        host_device_interface->float_val2 = float_param2;
      }
      if (use_double_param1) {
        host_device_interface->use_double_val1 = true;
        host_device_interface->double_val1 = double_param1;
      }
      if (use_double_param2) {
        host_device_interface->use_double_val2 = true;
        host_device_interface->double_val2 = double_param2;
      }
      if (use_str_param1) {
        str_param1_value = instance_getter_java_lang_String_value(gc_info, str_param1,
                          exception);
        str_param1_count = instance_getter_java_lang_String_count(gc_info, str_param1,
                          exception);
        
        for(int i = 0; i < str_param1_count; i++) {
          host_device_interface->str_val1[i] = char__array_get(gc_info, str_param1_value, i, exception);
        }
        host_device_interface->use_str_val1 = true;
        host_device_interface->str_val1[str_param1_count] = 
'\0'
;
      }
      if (use_str_param2) {
        str_param2_value = instance_getter_java_lang_String_value(gc_info, str_param2,
                           exception);
        str_param2_count = instance_getter_java_lang_String_count(gc_info, str_param2,
                           exception);
        
        for(int i = 0; i < str_param2_count; i++) {
          host_device_interface->str_val2[i] = char__array_get(gc_info, str_param2_value, i, exception);
        }
        host_device_interface->use_str_val2 = true;
        host_device_interface->str_val2[str_param2_count] = 
'\0'
;
      }
      if (use_str_param3) {
        str_param3_value = instance_getter_java_lang_String_value(gc_info, str_param3,
                           exception);
        str_param3_count = instance_getter_java_lang_String_count(gc_info, str_param3,
                           exception);
        
        for(int i = 0; i < str_param3_count; i++) {
          host_device_interface->str_val3[i] = char__array_get(gc_info, str_param3_value, i, exception);
        }
        host_device_interface->use_str_val3 = true;
        host_device_interface->str_val3[str_param3_count] = 
'\0'
;
      }
      
      host_device_interface->key_type = key_type;
      host_device_interface->value_type = value_type;
      /***********************************************************************/
      
      host_device_interface->has_task = true;
      __threadfence_system();
      
      /***********************************************************************/
      
      inner_timeout = 0;
      while (!host_device_interface->is_result_available) {
        __threadfence_system();
        
	
        if (++inner_timeout > 30000) {
          break;
        }
      }
      /***********************************************************************/
      
      if (return_type == HostDeviceInterface::KEY_VALUE_PAIR) {
        
        
        key_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, 
                      key_value_pair_ref, exception);
        key_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, key_obj_ref);
        
        if (key_type == HostDeviceInterface::INT) {
          *(( int *) &key_obj_deref[32]) = host_device_interface->int_val1;
        }
 else if (key_type == HostDeviceInterface::LONG) {
          *(( long long *) &key_obj_deref[32]) = host_device_interface->long_val1;
        }
 else if (key_type == HostDeviceInterface::FLOAT) {
          *(( float *) &key_obj_deref[32]) = host_device_interface->float_val1;
        }
 else if (key_type == HostDeviceInterface::DOUBLE) {
          *(( double *) &key_obj_deref[32]) = host_device_interface->double_val1;
        }
 else if (key_type == HostDeviceInterface::STRING) {
          int i;
          int len = at_illecker_strlen(host_device_interface->str_val1);
          int characters = char__array_new(gc_info, len, exception);
          for(i = 0; i < len; ++i) {
            char__array_set(gc_info, characters, i, host_device_interface->str_val1[i], exception);
          }
          
          *(( int *) &key_obj_deref[32]) = characters;
          
          *(( int *) &key_obj_deref[40]) = len;
          
          *(( int *) &key_obj_deref[44]) = 0;
        }
        
        value_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, 
                        key_value_pair_ref, exception);
        value_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, value_obj_ref);
        
        if (value_type == HostDeviceInterface::INT) {
          *(( int *) &value_obj_deref[32]) = host_device_interface->int_val2;
        }
 else if (value_type == HostDeviceInterface::LONG) {
          *(( long long *) &value_obj_deref[32]) = host_device_interface->long_val2;
        }
 else if (value_type == HostDeviceInterface::FLOAT) {
          *(( float *) &value_obj_deref[32]) = host_device_interface->float_val2;
        }
 else if (value_type == HostDeviceInterface::DOUBLE) {
          *(( double *) &value_obj_deref[32]) = host_device_interface->double_val2;
        }
 else if (value_type == HostDeviceInterface::STRING) {
          int i;
          int len = at_illecker_strlen(host_device_interface->str_val2);
          int characters = char__array_new(gc_info, len, exception);
          for(i = 0; i < len; ++i) {
            char__array_set(gc_info, characters, i, host_device_interface->str_val2[i], exception);
          }
          
          *(( int *) &value_obj_deref[32]) = characters;
          
          *(( int *) &value_obj_deref[40]) = len;
          
          *(( int *) &value_obj_deref[44]) = 0;
        }
        
        return_value = !host_device_interface->end_of_data;
      }
 else if (use_return_value) {
 
        
        if (return_type == HostDeviceInterface::INT) {
          return_value = host_device_interface->int_val1;
        }
 else if (return_type == HostDeviceInterface::LONG) {
          return_value = host_device_interface->long_val1;
        }
 else if (return_type == HostDeviceInterface::FLOAT) {
          return_value = host_device_interface->float_val1;
        }
 else if (return_type == HostDeviceInterface::DOUBLE) {
          return_value = host_device_interface->double_val1;
        }
 else if (return_type == HostDeviceInterface::STRING) {
          
          edu_syr_pcpratts_gc_assign(gc_info, (int*)&return_value,
            at_illecker_string_constant(gc_info, host_device_interface->str_val1, exception));
       
        }
 else if (return_type == HostDeviceInterface::STRING_ARRAY) {
          int index = 0;
          int array_len = host_device_interface->int_val1;
          if (array_len > 0) {
            
            return_value = java_lang_String__array_new(gc_info, array_len, exception);
            while ( (host_device_interface->use_int_val1) && (index < array_len) ) {
              if (host_device_interface->use_str_val1) {
                java_lang_String__array_set(gc_info, return_value, index, 
                  at_illecker_string_constant(gc_info, host_device_interface->str_val1, exception), exception);
                index++;
              }
              if (host_device_interface->use_str_val2) {
                java_lang_String__array_set(gc_info, return_value, index, 
                  at_illecker_string_constant(gc_info, host_device_interface->str_val2, exception), exception);
                index++;
              }
              if (host_device_interface->use_str_val3) {
                java_lang_String__array_set(gc_info, return_value, index, 
                  at_illecker_string_constant(gc_info, host_device_interface->str_val3, exception), exception);
                index++;
              }
              
              host_device_interface->is_result_available = false;
              __threadfence_system();
              
              while (!host_device_interface->is_result_available) {
                __threadfence_system();
              }
            }
          }
 else {
            return_value = 0;
          }
        }
      }
      /***********************************************************************/
      
      if ( (use_int_param1) || (return_type == HostDeviceInterface::INT) ) {
        host_device_interface->int_val1 = 0;
        host_device_interface->use_int_val1 = false;
      }
      if (use_int_param2) {
        host_device_interface->int_val2 = 0;
        host_device_interface->use_int_val2 = false;
      }
      if (use_int_param3) {
        host_device_interface->int_val3 = 0;
        host_device_interface->use_int_val3 = false;
      }
      if ( (use_long_param1) || (return_type == HostDeviceInterface::LONG) ) {
        host_device_interface->long_val1 = 0;
        host_device_interface->use_long_val1 = false;
      }
      if (use_long_param1) {
        host_device_interface->long_val2 = 0;
        host_device_interface->use_long_val2 = false;
      }
      if ( (use_float_param1) || (return_type == HostDeviceInterface::FLOAT) ) {
        host_device_interface->float_val1 = 0;
        host_device_interface->use_float_val1 = false;
      }
      if (use_float_param2) {
        host_device_interface->float_val2 = 0;
        host_device_interface->use_float_val2 = false;
      }
      if ( (use_double_param1) || (return_type == HostDeviceInterface::DOUBLE) ) {
        host_device_interface->double_val1 = 0;
        host_device_interface->use_double_val1 = false;
      }
      if (use_double_param2) {
        host_device_interface->double_val2 = 0;
        host_device_interface->use_double_val2 = false;
      }
      if ( (use_str_param1) || (return_type == HostDeviceInterface::STRING) ) {
        host_device_interface->str_val1[0] = 
'\0'
;
        host_device_interface->use_str_val1 = false;
      }
      if (use_str_param2) {
        host_device_interface->str_val2[0] = 
'\0'
;
        host_device_interface->use_str_val2 = false;
      }
      if (use_str_param3) {
        host_device_interface->str_val3[0] = 
'\0'
;
        host_device_interface->use_str_val3 = false;
      }
      if (return_type == HostDeviceInterface::STRING_ARRAY) {
        host_device_interface->int_val1 = 0;
        host_device_interface->use_int_val1 = false;
        host_device_interface->str_val1[0] = 
'\0'
;
        host_device_interface->use_str_val1 = false;
        host_device_interface->str_val2[0] = 
'\0'
;
        host_device_interface->use_str_val2 = false;
        host_device_interface->str_val3[0] = 
'\0'
;
        host_device_interface->use_str_val3 = false;
      }
      host_device_interface->command = HostDeviceInterface::UNDEFINED;
      host_device_interface->return_type = HostDeviceInterface::NOT_AVAILABLE;
      host_device_interface->key_type = HostDeviceInterface::NOT_AVAILABLE;
      host_device_interface->value_type = HostDeviceInterface::NOT_AVAILABLE;
      /***********************************************************************/ 
      
      host_device_interface->is_result_available = false;
      host_device_interface->lock_thread_id = -1;
      
      __threadfence_system();
      
      /***********************************************************************/ 
      
      done = true; 
    }
 else {
      count++;
      if (count > 50) {
        count = 0;
      }
    }
  }
  return return_value;
}
/*****************************************************************************/
/* Hama Peer public methods */
__device__
void edu_syr_pcpratts_rootbeer_runtime_HamaPeer_send( char * gc_info,
     int peer_name_str_ref, int message_obj_ref, int * exception) {
  int int_value = 0;
  bool use_int_value = false;
  long long long_value = 0;
  bool use_long_value = false;
  float float_value = 0;
  bool use_float_value = false;
  double double_value = 0;
  bool use_double_value = false;
  int string_value = 0;
  bool use_string_value = false;
  char * message_obj_deref;
  
  
  if (message_obj_ref == -1) {
    printf("Exception in HamaPeer.send: unsupported NULL Type\n");
    return;
  }
 else {
    
    if (at_illecker_typeof_Integer(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      int_value = *(( int *) &message_obj_deref[32]);
      use_int_value = true;
      
    }
 else if (at_illecker_typeof_Long(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      long_value = *(( long long *) &message_obj_deref[32]);
      use_long_value = true;
      
    }
 else if (at_illecker_typeof_Float(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      float_value = *(( float *) &message_obj_deref[32]);
      use_float_value = true;
      
    }
 else if (at_illecker_typeof_Double(gc_info, message_obj_ref)) {
      message_obj_deref = edu_syr_pcpratts_gc_deref(gc_info, message_obj_ref);
      double_value = *(( double *) &message_obj_deref[32]);
      use_double_value = true;
      
    }
 else if (at_illecker_typeof_String(gc_info, message_obj_ref)) {
      string_value = message_obj_ref;
      use_string_value = true;
      
    }
 else {
      
      printf("Exception in HamaPeer.send: unsupported Type\n");
      return;
    }
  }
  at_illecker_getResult<int>(gc_info, HostDeviceInterface::SEND_MSG,
    HostDeviceInterface::NOT_AVAILABLE, false, 
    0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
    int_value, use_int_value,
    0, false,
    0, false,
    long_value, use_long_value,
    0, false,
    float_value, use_float_value,
    0, false,
    double_value, use_double_value,
    0, false,
    peer_name_str_ref, true,
    string_value, use_string_value,
    0, false,
    exception);
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getCurrentStringMessage( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::GET_MSG,
           HostDeviceInterface::STRING, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getNumCurrentMessages( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::GET_MSG_COUNT,
           HostDeviceInterface::INT, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
void edu_syr_pcpratts_rootbeer_runtime_HamaPeer_sync( char * gc_info, 
     int * exception) {
  at_illecker_getResult<int>(gc_info, HostDeviceInterface::SYNC,
    HostDeviceInterface::NOT_AVAILABLE, false, 
    0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    0, false,
    exception);
}
__device__
long edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getSuperstepCount( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<long>(gc_info, HostDeviceInterface::GET_SUPERSTEP_COUNT,
           HostDeviceInterface::LONG, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
int edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getAllPeerNames( char * gc_info, 
    int * exception) {
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::GET_ALL_PEERNAME,
           HostDeviceInterface::STRING_ARRAY, true, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
void edu_syr_pcpratts_rootbeer_runtime_HamaPeer_reopenInput( char * gc_info, 
     int * exception) {
  at_illecker_getResult<int>(gc_info, HostDeviceInterface::REOPEN_INPUT,
           HostDeviceInterface::NOT_AVAILABLE, false, 
           0, HostDeviceInterface::NOT_AVAILABLE, HostDeviceInterface::NOT_AVAILABLE,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__
bool edu_syr_pcpratts_rootbeer_runtime_HamaPeer_readNext( char * gc_info, 
     int key_value_pair_ref, int * exception) {
  int key_obj_ref;
  int value_obj_ref;
  HostDeviceInterface::TYPE key_type;
  HostDeviceInterface::TYPE value_type;
  key_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, 
                key_value_pair_ref, exception);
  value_obj_ref = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, 
                  key_value_pair_ref, exception);
  
  if (at_illecker_typeof_Integer(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::INT;
  }
 else if (at_illecker_typeof_Long(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::LONG;
  }
 else if (at_illecker_typeof_Float(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::FLOAT;
  }
 else if (at_illecker_typeof_Double(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::DOUBLE;
  }
 else if (at_illecker_typeof_String(gc_info, key_obj_ref)) {
    key_type = HostDeviceInterface::STRING;
  }
 else if (key_obj_ref == -1) {
    key_type = HostDeviceInterface::NULL_TYPE;
  }
 else {
    
    printf("Exception in HamaPeer.readNext: unsupported Key Type\n");
    return false;
  }
  
  if (at_illecker_typeof_Integer(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::INT;
  }
 else if (at_illecker_typeof_Long(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::LONG;
  }
 else if (at_illecker_typeof_Float(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::FLOAT;
  }
 else if (at_illecker_typeof_Double(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::DOUBLE;
  }
 else if (at_illecker_typeof_String(gc_info, value_obj_ref)) {
    value_type = HostDeviceInterface::STRING;
  }
 else if (value_obj_ref == -1) {
    value_type = HostDeviceInterface::NULL_TYPE;
  }
 else {
    
    printf("Exception in HamaPeer.readNext: unsupported Value Type\n");
    return false;
  }
  if ( (key_type == HostDeviceInterface::NULL_TYPE) &&
       (value_type == HostDeviceInterface::NULL_TYPE) ) {
    printf("Exception in HamaPeer.readNext: key and value are NULL!\n");
    return false;
  }
  return at_illecker_getResult<int>(gc_info, HostDeviceInterface::READ_KEYVALUE,
           HostDeviceInterface::KEY_VALUE_PAIR, false, 
           key_value_pair_ref, key_type, value_type,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           0, false,
           exception);
}
__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_getKey13_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $r1 = -1;
 r0  =  thisref ;
 $r1  = instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
return  $r1 ;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_getNearestCenter5_a12_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int i0;
double d0;
int i1;
double d1;
int $r2 = -1;
int $i2;
int $r3 = -1;
int $r4 = -1;
char $b3;
 r0  =  thisref ;
 r1  =  parameter0 ;
 i0  =  0 ;
 d0  =  1.7976931348623157E308 ;
 i1  =  0 ;
label2:
 $r2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $i2  = edu_syr_pcpratts_array_length(gc_info,  $r2 );
if ( i1  >=  $i2   ) goto label0;
 $r3  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $r4  = double__array__array_get(gc_info, $r3, i1, exception);
if(*exception != 0) {
 
return 0; }
 d1  = at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_measureEuclidianDistance8_a12_a12_(gc_info,
 r0,
  $r4 ,
  r1 , exception);
if(*exception != 0) {
 
return 0; }
 $b3  = edu_syr_pcpratts_cmpg((double) d1 , (double) d0 );
if ( $b3  >=  0   ) goto label1;
 d0  =  d1 ;
 i0  =  i1 ;
label1:
 i1  =  i1  +  1  ;
goto label2;
label0:
return  i0 ;
  return 0;
}
__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int r1 = -1;
int i0;
int i1;
int $r2 = -1;
int $r3 = -1;
int $i2;
int $i3;
int $r4 = -1;
int $i4;
int $i5;
int $r5 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 64));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2905);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 64);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_String_value(gc_info, thisref, -1, exception);
instance_setter_java_lang_String_count(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_hash(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_offset(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
 i0  =  parameter1 ;
 i1  =  parameter2 ;
if ( i0  >=  0   ) goto label0;
 $r2  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r2, java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_(gc_info,
  i0 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r2 ;
return 0;
label0:
if ( i1  >=  0   ) goto label1;
 $r3  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r3, java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_(gc_info,
  i1 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r3 ;
return 0;
label1:
 $i2  = edu_syr_pcpratts_array_length(gc_info,  r1 );
 $i3  =  $i2  -  i1  ;
if ( i0  <=  $i3   ) goto label2;
 $r4  =  -1 ;
 $i4  =  i0  +  i1  ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r4, java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_(gc_info,
  $i4 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r4 ;
return 0;
label2:
instance_setter_java_lang_String_offset(gc_info, r0,  0 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_count(gc_info, r0,  i1 , exception);
if(*exception != 0) {
 
return 0; }
 $i5  =  i0  +  i1  ;
 $r5  = java_util_Arrays_copyOfRangea14_a14_5_5_(gc_info,  r1 ,  i0 ,  $i5 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_value(gc_info, r0,  $r5 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_gpuMethod0_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int i3;
int i4;
int i5;
int i6;
char z0;
int i7;
int r1 = -1;
int i8;
int r2 = -1;
int r3 = -1;
int r4 = -1;
int i9;
int r5 = -1;
int i10;
int r6 = -1;
int i11;
int r7 = -1;
int i12;
int $i13;
int $r8 = -1;
int $i14;
int $r9 = -1;
int $r10 = -1;
int $i15;
int $r11 = -1;
int $r12 = -1;
int $i16;
int $r13 = -1;
int $r14 = -1;
int $i17;
int $r15 = -1;
char z1;
int $r16 = -1;
int $r17 = -1;
int $r18 = -1;
int $r19 = -1;
int $r20 = -1;
int r21 = -1;
int $r22 = -1;
int $i18;
int $r23 = -1;
int $r24 = -1;
int i19;
int $r25 = -1;
int r26 = -1;
int $i20;
int $r27 = -1;
int $r28 = -1;
int $i21;
int r29 = -1;
int i22;
int r30 = -1;
int i23;
int $r31 = -1;
int $i24;
int $r32 = -1;
int $i25;
int r33 = -1;
int $r34 = -1;
int $r35 = -1;
int $r36 = -1;
int $r37 = -1;
int r38 = -1;
int $r39 = -1;
int $r40 = -1;
int $r41 = -1;
int r42 = -1;
int $r43 = -1;
int $r44 = -1;
int $r45 = -1;
int $i26;
int $r46 = -1;
int $r47 = -1;
int r48 = -1;
int $r49 = -1;
int $r50 = -1;
int $r51 = -1;
int r52 = -1;
int i27;
int $r53 = -1;
int $r54 = -1;
int $i28;
int $r55 = -1;
int $r56 = -1;
int $r57 = -1;
int $r58 = -1;
double $d0;
int $r59 = -1;
int $r60 = -1;
int $r61 = -1;
int $r62 = -1;
int $i29;
int $i30;
int $r63 = -1;
int $r64 = -1;
int $r65 = -1;
int $r66 = -1;
int $r67 = -1;
int $r68 = -1;
int r69 = -1;
int i31;
int $r70 = -1;
int $i32;
int $r71 = -1;
int $r72 = -1;
int $i33;
int r73 = -1;
int $r74 = -1;
int $i34;
int r75 = -1;
int i35;
int i36;
int r76 = -1;
int $r77 = -1;
int $r78 = -1;
int $r79 = -1;
int r80 = -1;
int $r81 = -1;
int i37;
int $r82 = -1;
int $r83 = -1;
int $r84 = -1;
double $d1;
int $i38;
int i39;
int $r85 = -1;
int $i40;
int $r86 = -1;
double $d2;
double $d3;
double $d4;
int $i43;
int $i44;
int i45;
int $i46;
int $i47;
int i48;
int $r88 = -1;
int $i49;
int $r89 = -1;
double $d5;
int $i51;
double $d6;
double $d7;
long long l52;
int i53;
int $i54;
int $i55;
double d8;
int i56;
int $r90 = -1;
int $r91 = -1;
int $i57;
int $r92 = -1;
int $r93 = -1;
double $d9;
int $r94 = -1;
double $d10;
double $d11;
double $d12;
int $r95 = -1;
int $r96 = -1;
char $b58;
int $r97 = -1;
int $r98 = -1;
long long $l59;
int $r99 = -1;
int $r100 = -1;
long long $l60;
long long $l61;
char $b62;
int $i63;
int $i64;
long long $l65;
long long $l66;
char $b67;
int $r101 = -1;
int $r102 = -1;
 r0  =  thisref ;
 i0  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getBlockDimx(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i1  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getGridDimx(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i2  =  i0  *  i1  ;
 i3  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getBlockIdxx(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i4  = edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_getThreadIdxx(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $i13  =  i3  *  i0  ;
 i5  =  $i13  +  i4  ;
label55:
if ( i5  !=  0   ) goto label0;
 $r8  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i14  = edu_syr_pcpratts_array_length(gc_info,  $r8 );
 $r9  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r10  = double__array__array_get(gc_info, $r9, 0, exception);
if(*exception != 0) {
 
return ; }
 $i15  = edu_syr_pcpratts_array_length(gc_info,  $r10 );
 $r11  = double__array__array_new_multi_array(gc_info,  $i14 ,  $i15 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0,  $r11 , exception);
if(*exception != 0) {
 
return ; }
 $r12  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i16  = edu_syr_pcpratts_array_length(gc_info,  $r12 );
 $r13  = int__array_new(gc_info,  $i16 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0,  $r13 , exception);
if(*exception != 0) {
 
return ; }
 i6  =  0 ;
label2:
 $r14  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i17  = edu_syr_pcpratts_array_length(gc_info,  $r14 );
if ( i6  >=  $i17   ) goto label0;
 $r15  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
int__array_set(gc_info, $r15, i6,  -1 , exception);
if(*exception != 0) {
 
return ; }
 i6  =  i6  +  1  ;
goto label2;
label0:
 z1  =  1 ;
 z0  =  0 ;
 i7  =  0 ;
label20:
if ( z1  ==  0   ) goto label3;
 r1  = (int)  -1 ;
 i8  =  0 ;
if ( i5  !=  0   ) goto label4;
 $r16  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( $r16  ==  -1   ) goto label5;
if ( z0  ==  0   ) goto label6;
label5:
 $r17  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( $r17  !=  -1   ) goto label7;
 $r18  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r18, at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache(gc_info, r0,  $r18 , exception);
if(*exception != 0) {
 
return ; }
 z0  =  1 ;
label7:
 r2  =  edu_syr_pcpratts_string_constant(gc_info, (char *) "", exception) ;
 $r19  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r19, edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a660_13_13_(gc_info,
  r2 ,
  -1 , exception));
if(*exception != 0) {
 
return ; }
 r3  =  $r19 ;
label12:
if ( i8  >=  i2   ) goto label8;
 z1  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_readNext(gc_info,  r3 , exception);
if(*exception != 0) {
 
return ; }
 z0  =  z1 ;
if ( z1  !=  0   ) goto label9;
goto label8;
label9:
 $r20  = edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_getKey13_(gc_info,
 r3, exception);
if(*exception != 0) {
 
return ; }
 r21  = (int)  $r20 ;
 $r22  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r22, at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a660_9_(gc_info,
  r21 , exception));
if(*exception != 0) {
 
return ; }
 r4  =  $r22 ;
if ( r1  !=  -1   ) goto label11;
 $i18  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_getLength5_(gc_info,
 r4, exception);
if(*exception != 0) {
 
return ; }
 r1  = double__array__array_new_multi_array(gc_info,  i2 ,  $i18 , exception);
label11:
 $r23  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_(gc_info,
 r4, exception);
if(*exception != 0) {
 
return ; }
double__array__array_set(gc_info, r1, i8,  $r23 , exception);
if(*exception != 0) {
 
return ; }
 $r24  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_add0_11_(gc_info,
 $r24,
  r4 , exception);
if(*exception != 0) {
 
return ; }
 i8  =  i8  +  1  ;
goto label12;
label8:
goto label4;
label6:
 i19  =  i7 ;
label18:
if ( i8  >=  i2   ) goto label14;
 $r25  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 r26  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_get11_5_(gc_info,
 $r25,
  i19 , exception);
if(*exception != 0) {
 
return ; }
if ( r1  !=  -1   ) goto label15;
 $i20  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_getLength5_(gc_info,
 r26, exception);
if(*exception != 0) {
 
return ; }
 r1  = double__array__array_new_multi_array(gc_info,  i2 ,  $i20 , exception);
label15:
 $r27  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_(gc_info,
 r26, exception);
if(*exception != 0) {
 
return ; }
double__array__array_set(gc_info, r1, i8,  $r27 , exception);
if(*exception != 0) {
 
return ; }
 i8  =  i8  +  1  ;
 i19  =  i19  +  1  ;
 $r28  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i21  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_getLength5_(gc_info,
 $r28, exception);
if(*exception != 0) {
 
return ; }
if ( i19  !=  $i21   ) goto label16;
 z1  =  0 ;
goto label14;
label16:
goto label18;
label14:
 i7  =  i19 ;
label4:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
if ( i5  >=  i8   ) goto label19;
 r29  = double__array__array_get(gc_info, r1, i5, exception);
if(*exception != 0) {
 
return ; }
 i22  = at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_getNearestCenter5_a12_(gc_info,
 r0,
  r29 , exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_assignCenters0_5_a12_(gc_info,
 r0,
  i22 ,
  r29 , exception);
if(*exception != 0) {
 
return ; }
label19:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
goto label20;
label3:
if ( i5  !=  0   ) goto label21;
 r30  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getAllPeerNames(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i23  =  0 ;
label29:
 $r31  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i24  = edu_syr_pcpratts_array_length(gc_info,  $r31 );
if ( i23  >=  $i24   ) goto label21;
 $r32  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i25  = int__array_get(gc_info, $r32, i23, exception);
if(*exception != 0) {
 
return ; }
if ( $i25  ==  -1   ) goto label23;
 r33  =  edu_syr_pcpratts_string_constant(gc_info, (char *) "", exception) ;
 $r34  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r34, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r35  = java_lang_StringBuilder_append10_9_(gc_info,
 $r34,
  r33 , exception);
if(*exception != 0) {
 
return ; }
 $r36  = java_lang_Integer_toString9_5_(gc_info,  i23 , exception);
if(*exception != 0) {
 
return ; }
 $r37  = java_lang_StringBuilder_append10_9_(gc_info,
 $r35,
  $r36 , exception);
if(*exception != 0) {
 
return ; }
 r38  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r37, exception);
if(*exception != 0) {
 
return ; }
 $r39  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r39, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r40  = java_lang_StringBuilder_append10_9_(gc_info,
 $r39,
  r38 , exception);
if(*exception != 0) {
 
return ; }
 $r41  = java_lang_StringBuilder_append10_9_(gc_info,
 $r40,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ":", exception) , exception);
if(*exception != 0) {
 
return ; }
 r42  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r41, exception);
if(*exception != 0) {
 
return ; }
 $r43  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r43, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r44  = java_lang_StringBuilder_append10_9_(gc_info,
 $r43,
  r42 , exception);
if(*exception != 0) {
 
return ; }
 $r45  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i26  = int__array_get(gc_info, $r45, i23, exception);
if(*exception != 0) {
 
return ; }
 $r46  = java_lang_Integer_toString9_5_(gc_info,  $i26 , exception);
if(*exception != 0) {
 
return ; }
 $r47  = java_lang_StringBuilder_append10_9_(gc_info,
 $r44,
  $r46 , exception);
if(*exception != 0) {
 
return ; }
 r48  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r47, exception);
if(*exception != 0) {
 
return ; }
 $r49  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r49, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r50  = java_lang_StringBuilder_append10_9_(gc_info,
 $r49,
  r48 , exception);
if(*exception != 0) {
 
return ; }
 $r51  = java_lang_StringBuilder_append10_9_(gc_info,
 $r50,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ":", exception) , exception);
if(*exception != 0) {
 
return ; }
 r52  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r51, exception);
if(*exception != 0) {
 
return ; }
 i27  =  0 ;
label26:
 $r53  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r54  = double__array__array_get(gc_info, $r53, i23, exception);
if(*exception != 0) {
 
return ; }
 $i28  = edu_syr_pcpratts_array_length(gc_info,  $r54 );
if ( i27  >=  $i28   ) goto label24;
 $r55  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r55, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r56  = java_lang_StringBuilder_append10_9_(gc_info,
 $r55,
  r52 , exception);
if(*exception != 0) {
 
return ; }
 $r57  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r58  = double__array__array_get(gc_info, $r57, i23, exception);
if(*exception != 0) {
 
return ; }
 $d0  = double__array_get(gc_info, $r58, i27, exception);
if(*exception != 0) {
 
return ; }
 $r59  = java_lang_Double_toString9_8_(gc_info,  $d0 , exception);
if(*exception != 0) {
 
return ; }
 $r60  = java_lang_StringBuilder_append10_9_(gc_info,
 $r56,
  $r59 , exception);
if(*exception != 0) {
 
return ; }
 r52  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r60, exception);
if(*exception != 0) {
 
return ; }
 $r61  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r62  = double__array__array_get(gc_info, $r61, i23, exception);
if(*exception != 0) {
 
return ; }
 $i29  = edu_syr_pcpratts_array_length(gc_info,  $r62 );
 $i30  =  $i29  -  1  ;
if ( i27  >=  $i30   ) goto label25;
 $r63  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r63, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return ; }
 $r64  = java_lang_StringBuilder_append10_9_(gc_info,
 $r63,
  r52 , exception);
if(*exception != 0) {
 
return ; }
 $r65  = java_lang_StringBuilder_append10_9_(gc_info,
 $r64,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ", ", exception) , exception);
if(*exception != 0) {
 
return ; }
 r52  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r65, exception);
if(*exception != 0) {
 
return ; }
label25:
 i27  =  i27  +  1  ;
goto label26;
label24:
 $r66  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r66,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "send message: \'", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r67  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r67,
  r52 , exception);
if(*exception != 0) {
 
return ; }
 $r68  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r68,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "\'", exception) , exception);
if(*exception != 0) {
 
return ; }
 r69  =  r30 ;
 i31  = edu_syr_pcpratts_array_length(gc_info,  r69 );
 i9  =  0 ;
label28:
if ( i9  >=  i31   ) goto label23;
 r5  = java_lang_String__array_get(gc_info, r69, i9, exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_HamaPeer_send(gc_info,  r5 ,  r52 , exception);
if(*exception != 0) {
 
return ; }
 i9  =  i9  +  1  ;
goto label28;
label23:
 i23  =  i23  +  1  ;
goto label29;
label21:
edu_syr_pcpratts_rootbeer_runtime_HamaPeer_sync(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $r70  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i32  = edu_syr_pcpratts_array_length(gc_info,  $r70 );
 $r71  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r72  = double__array__array_get(gc_info, $r71, 0, exception);
if(*exception != 0) {
 
return ; }
 $i33  = edu_syr_pcpratts_array_length(gc_info,  $r72 );
 r73  = double__array__array_new_multi_array(gc_info,  $i32 ,  $i33 , exception);
 $r74  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i34  = edu_syr_pcpratts_array_length(gc_info,  $r74 );
 r75  = int__array_new(gc_info,  $i34 , exception);
if ( i5  !=  0   ) goto label30;
 i35  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getNumCurrentMessages(gc_info, exception);
if(*exception != 0) {
 
return ; }
 i36  =  0 ;
label38:
if ( i36  >=  i35   ) goto label31;
 r76  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getCurrentStringMessage(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $r77  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r77,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "got message: \'", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r78  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r78,
  r76 , exception);
if(*exception != 0) {
 
return ; }
 $r79  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r79,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "\'", exception) , exception);
if(*exception != 0) {
 
return ; }
 r80  = java_lang_String_split(gc_info,
 r76,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ":", exception) ,
  3 , exception);
if(*exception != 0) {
 
return ; }
 $r81  = java_lang_String__array_get(gc_info, r80, 0, exception);
if(*exception != 0) {
 
return ; }
 i37  = java_lang_Integer_parseInt(gc_info,  $r81 , exception);
if(*exception != 0) {
 
return ; }
 $r82  = java_lang_String__array_get(gc_info, r80, 1, exception);
if(*exception != 0) {
 
return ; }
 i10  = java_lang_Integer_parseInt(gc_info,  $r82 , exception);
if(*exception != 0) {
 
return ; }
 $r83  = java_lang_String__array_get(gc_info, r80, 2, exception);
if(*exception != 0) {
 
return ; }
 r6  = java_lang_String_split(gc_info,
 $r83,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ",", exception) , exception);
if(*exception != 0) {
 
return ; }
 i11  = edu_syr_pcpratts_array_length(gc_info,  r6 );
 r7  = double__array_new(gc_info,  i11 , exception);
 i12  =  0 ;
label33:
if ( i12  >=  i11   ) goto label32;
 $r84  = java_lang_String__array_get(gc_info, r6, i12, exception);
if(*exception != 0) {
 
return ; }
 $d1  = java_lang_Double_parseDouble(gc_info,  $r84 , exception);
if(*exception != 0) {
 
return ; }
double__array_set(gc_info, r7, i12,  $d1 , exception);
if(*exception != 0) {
 
return ; }
 i12  =  i12  +  1  ;
goto label33;
label32:
 $i38  = int__array_get(gc_info, r75, i37, exception);
if(*exception != 0) {
 
return ; }
if ( $i38  !=  0   ) goto label34;
double__array__array_set(gc_info, r73, i37,  r7 , exception);
if(*exception != 0) {
 
return ; }
goto label35;
label34:
 i39  =  0 ;
label37:
 $r85  = double__array__array_get(gc_info, r73, i37, exception);
if(*exception != 0) {
 
return ; }
 $i40  = edu_syr_pcpratts_array_length(gc_info,  $r85 );
if ( i39  >=  $i40   ) goto label35;
 $r86  = double__array__array_get(gc_info, r73, i37, exception);
if(*exception != 0) {
 
return ; }
 $d2  = double__array_get(gc_info, $r86, i39, exception);
if(*exception != 0) {
 
return ; }
 $d3  = double__array_get(gc_info, r7, i39, exception);
if(*exception != 0) {
 
return ; }
 $d4  =  $d2  +  $d3  ;
double__array_set(gc_info, $r86, i39,  $d4 , exception);
if(*exception != 0) {
 
return ; }
 i39  =  i39  +  1  ;
goto label37;
label35:
 $i43  = int__array_get(gc_info, r75, i37, exception);
if(*exception != 0) {
 
return ; }
 $i44  =  $i43  +  i10  ;
int__array_set(gc_info, r75, i37,  $i44 , exception);
if(*exception != 0) {
 
return ; }
 i36  =  i36  +  1  ;
goto label38;
label31:
 i45  =  0 ;
label43:
 $i46  = edu_syr_pcpratts_array_length(gc_info,  r73 );
if ( i45  >=  $i46   ) goto label39;
 $i47  = int__array_get(gc_info, r75, i45, exception);
if(*exception != 0) {
 
return ; }
if ( $i47  ==  0   ) goto label40;
 i48  =  0 ;
label42:
 $r88  = double__array__array_get(gc_info, r73, i45, exception);
if(*exception != 0) {
 
return ; }
 $i49  = edu_syr_pcpratts_array_length(gc_info,  $r88 );
if ( i48  >=  $i49   ) goto label40;
 $r89  = double__array__array_get(gc_info, r73, i45, exception);
if(*exception != 0) {
 
return ; }
 $d5  = double__array_get(gc_info, $r89, i48, exception);
if(*exception != 0) {
 
return ; }
 $i51  = int__array_get(gc_info, r75, i45, exception);
if(*exception != 0) {
 
return ; }
 $d6  = (double)  $i51 ;
 $d7  =  $d5  /  $d6  ;
double__array_set(gc_info, $r89, i48,  $d7 , exception);
if(*exception != 0) {
 
return ; }
 i48  =  i48  +  1  ;
goto label42;
label40:
 i45  =  i45  +  1  ;
goto label43;
label39:
 l52  =  0L ;
 i53  =  0 ;
label49:
 $i54  = edu_syr_pcpratts_array_length(gc_info,  r73 );
if ( i53  >=  $i54   ) goto label44;
 $i55  = int__array_get(gc_info, r75, i53, exception);
if(*exception != 0) {
 
return ; }
if ( $i55  ==  0   ) goto label45;
 d8  =  0.0 ;
 i56  =  0 ;
label47:
 $r90  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r91  = double__array__array_get(gc_info, $r90, i53, exception);
if(*exception != 0) {
 
return ; }
 $i57  = edu_syr_pcpratts_array_length(gc_info,  $r91 );
if ( i56  >=  $i57   ) goto label46;
 $r92  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r93  = double__array__array_get(gc_info, $r92, i53, exception);
if(*exception != 0) {
 
return ; }
 $d9  = double__array_get(gc_info, $r93, i56, exception);
if(*exception != 0) {
 
return ; }
 $r94  = double__array__array_get(gc_info, r73, i53, exception);
if(*exception != 0) {
 
return ; }
 $d10  = double__array_get(gc_info, $r94, i56, exception);
if(*exception != 0) {
 
return ; }
 $d11  =  $d9  -  $d10  ;
 $d12  = java_lang_Math_abs8_8_(gc_info,  $d11 , exception);
if(*exception != 0) {
 
return ; }
 d8  =  d8  +  $d12  ;
 i56  =  i56  +  1  ;
goto label47;
label46:
 $r95  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r95,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "calculateError: ", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r96  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_8_(gc_info,
 $r96,
  d8 , exception);
if(*exception != 0) {
 
return ; }
 $b58  = edu_syr_pcpratts_cmpl((double) d8 , (double) 0.0 );
if ( $b58  <=  0   ) goto label45;
 $r97  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r98  = double__array__array_get(gc_info, r73, i53, exception);
if(*exception != 0) {
 
return ; }
double__array__array_set(gc_info, $r97, i53,  $r98 , exception);
if(*exception != 0) {
 
return ; }
 l52  =  l52  +  1L  ;
label45:
 i53  =  i53  +  1  ;
goto label49;
label44:
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged(gc_info, r0,  l52 , exception);
if(*exception != 0) {
 
return ; }
 $l59  = edu_syr_pcpratts_rootbeer_runtime_HamaPeer_getSuperstepCount(gc_info, exception);
if(*exception != 0) {
 
return ; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_superstepCount(gc_info, r0,  $l59 , exception);
if(*exception != 0) {
 
return ; }
edu_syr_pcpratts_rootbeer_runtime_HamaPeer_reopenInput(gc_info, exception);
if(*exception != 0) {
 
return ; }
label30:
edu_syr_pcpratts_rootbeer_runtime_RootbeerGpu_syncthreads(gc_info, exception);
if(*exception != 0) {
 
return ; }
 $r99  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_print0_9_(gc_info,
 $r99,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "m_converged: ", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r100  = static_getter_java_lang_System_out(gc_info, exception);
 $l60  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
java_io_PrintStream_println0_6_(gc_info,
 $r100,
  $l60 , exception);
if(*exception != 0) {
 
return ; }
 $l61  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $b62  = edu_syr_pcpratts_cmp( $l61 ,  0L );
if ( $b62  !=  0   ) goto label50;
goto label51;
label50:
 $i63  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_maxIterations(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( $i63  <=  0   ) goto label52;
 $i64  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_maxIterations(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $l65  = (long long)  $i64 ;
 $l66  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_superstepCount(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $b67  = edu_syr_pcpratts_cmp( $l65 ,  $l66 );
if ( $b67  >=  0   ) goto label52;
goto label51;
label52:
goto label55;
label51:
 $r101  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r101,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "Finished! Writing the assignments...", exception) , exception);
if(*exception != 0) {
 
return ; }
 $r102  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r102,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "Done.", exception) , exception);
if(*exception != 0) {
 
return ; }
return;
}
__device__ int int__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ void int__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int int__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 3346);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  int__array_set(gc_info, thisref, i, 0, exception);
}
return thisref;
}
__device__ void java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
int i0;
int $i1;
int $r3 = -1;
int $i2;
int $r4 = -1;
int $r5 = -1;
double $d0;
int $i3;
int $r6 = -1;
int $r7 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2907);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, thisref, -1, exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
 r2  = java_lang_String_split(gc_info,
 r1,
  edu_syr_pcpratts_string_constant(gc_info, (char *) ",", exception) , exception);
if(*exception != 0) {
 
return 0; }
if ( r2  ==  -1   ) goto label0;
 $i1  = edu_syr_pcpratts_array_length(gc_info,  r2 );
 $r3  = double__array_new(gc_info,  $i1 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0,  $r3 , exception);
if(*exception != 0) {
 
return 0; }
 i0  =  0 ;
label2:
 $i2  = edu_syr_pcpratts_array_length(gc_info,  r2 );
if ( i0  >=  $i2   ) goto label1;
 $r4  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $r5  = java_lang_String__array_get(gc_info, r2, i0, exception);
if(*exception != 0) {
 
return 0; }
 $d0  = java_lang_Double_parseDouble(gc_info,  $r5 , exception);
if(*exception != 0) {
 
return 0; }
double__array_set(gc_info, $r4, i0,  $d0 , exception);
if(*exception != 0) {
 
return 0; }
 i0  =  i0  +  1  ;
goto label2;
label1:
 $i3  = edu_syr_pcpratts_array_length(gc_info,  r2 );
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0,  $i3 , exception);
if(*exception != 0) {
 
return 0; }
goto label3;
label0:
 $r6  = static_getter_java_lang_System_out(gc_info, exception);
java_io_PrintStream_println0_9_(gc_info,
 $r6,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "DenseDoubleVector no values found!", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $r7  = double__array_new(gc_info,  128 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0,  $r7 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0,  0 , exception);
if(*exception != 0) {
 
return 0; }
label3:
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_toArraya12_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int r1 = -1;
int i0;
int $i1;
int $i2;
int $r2 = -1;
double $d0;
 r0  =  thisref ;
 $i1  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 r1  = double__array_new(gc_info,  $i1 , exception);
 i0  =  0 ;
label1:
 $i2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
if ( i0  >=  $i2   ) goto label0;
 $r2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $d0  = double__array_get(gc_info, $r2, i0, exception);
if(*exception != 0) {
 
return 0; }
double__array_set(gc_info, r1, i0,  $d0 , exception);
if(*exception != 0) {
 
return 0; }
 i0  =  i0  +  1  ;
goto label1;
label0:
return  r1 ;
  return 0;
}
__device__ double at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_measureEuclidianDistance8_a12_a12_( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
double d0;
int i0;
int i1;
double d1;
double $d2;
double $d3;
double $d4;
double $d5;
 r0  =  thisref ;
 r1  =  parameter0 ;
 r2  =  parameter1 ;
 d0  =  0.0 ;
 i0  = edu_syr_pcpratts_array_length(gc_info,  r1 );
 i1  =  0 ;
label1:
if ( i1  >=  i0   ) goto label0;
 $d2  = double__array_get(gc_info, r2, i1, exception);
if(*exception != 0) {
 
return 0; }
 $d3  = double__array_get(gc_info, r1, i1, exception);
if(*exception != 0) {
 
return 0; }
 d1  =  $d2  -  $d3  ;
 $d4  =  d1  *  d1  ;
 d0  =  d0  +  $d4  ;
 i1  =  i1  +  1  ;
goto label1;
label0:
 $d5  = java_lang_Math_sqrt8_8_(gc_info,  d0 , exception);
if(*exception != 0) {
 
return 0; }
return  $d5 ;
  return 0;
}
__device__ int instance_getter_java_lang_String_hash( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[44]);
}
__device__ void instance_setter_java_lang_String_hash( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[44]) = parameter0;
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_java_lang_AbstractStringBuilder_count( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int static_getter_java_lang_System_out( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[12]);
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_cache( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[36]) = parameter0;
}
__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[36]);
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[36]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_java_lang_Class_name( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_vector( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( long long *) &thisref_deref[48]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_converged( char * gc_info, int thisref, long long parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( long long *) &thisref_deref[48]) = parameter0;
}
__device__ int static_getter_java_lang_Integer_sizeTable( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[8]);
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[44]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[44]) = parameter0;
}
__device__ int static_getter_java_lang_Integer_DigitOnes( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[16]);
}
__device__ long long instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_superstepCount( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( long long *) &thisref_deref[56]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_superstepCount( char * gc_info, int thisref, long long parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( long long *) &thisref_deref[56]) = parameter0;
}
__device__ int static_getter_java_lang_Integer_DigitTens( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[20]);
}
__device__ void instance_setter_java_lang_Throwable_cause( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_maxIterations( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[64]);
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_centers( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[36]);
}
__device__ int instance_getter_java_lang_String_count( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_java_lang_String_count( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_java_lang_AbstractStringBuilder_value( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[40]);
}
__device__ void instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[40]) = parameter0;
}
__device__ int instance_getter_java_lang_String_value( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[32]);
}
__device__ void instance_setter_java_lang_String_value( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[32]) = parameter0;
}
__device__ int instance_getter_java_lang_String_offset( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
return *(( int *) &thisref_deref[48]);
}
__device__ void instance_setter_java_lang_String_offset( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[48]) = parameter0;
}
__device__ void instance_setter_java_lang_Throwable_detailMessage( char * gc_info, int thisref, int parameter0, int * exception){
 char * thisref_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
*(( int *) &thisref_deref[36]) = parameter0;
}
__device__ int static_getter_java_lang_Integer_digits( char * gc_info, int * exception){
 char * thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, 0);
return *(( int *) &thisref_deref[0]);
}
__device__ void java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_Exception_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return;
}
__device__ int java_lang_Integer_toString9_5_( char * gc_info, int parameter0, int * exception){
int i0;
int i1;
int r0 = -1;
int $i2;
int $i3;
int $i4;
int $r1 = -1;
 i0  =  parameter0 ;
if ( i0  !=  -2147483648   ) goto label0;
return  edu_syr_pcpratts_string_constant(gc_info, (char *) "-2147483648", exception) ;
label0:
if ( i0  >=  0   ) goto label1;
 $i2  = - i0 ;
 $i3  = java_lang_Integer_stringSize5_5_(gc_info,  $i2 , exception);
if(*exception != 0) {
 
return 0; }
 $i4  =  $i3  +  1  ;
goto label2;
label1:
 $i4  = java_lang_Integer_stringSize5_5_(gc_info,  i0 , exception);
if(*exception != 0) {
 
return 0; }
label2:
 i1  =  $i4 ;
 r0  = char__array_new(gc_info,  i1 , exception);
java_lang_Integer_getChars0_5_5_a14_(gc_info,  i0 ,  i1 ,  r0 , exception);
if(*exception != 0) {
 
return 0; }
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_String_initab850b60f96d11de8a390800200c9a660_5_5_a14_(gc_info,
  0 ,
  i1 ,
  r0 , exception));
if(*exception != 0) {
 
return 0; }
return  $r1 ;
  return 0;
}
__device__ int java_lang_Integer_toUnsignedString9_5_5_( char * gc_info, int parameter0, int parameter1, int * exception){
int i0;
int i1;
int r0 = -1;
int i2;
int i3;
int i4;
int $r1 = -1;
int $i5;
char $c6;
int $r2 = -1;
int $i7;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 r0  = char__array_new(gc_info,  32 , exception);
 i2  =  32 ;
 i3  =  1  <<  i1  ;
 i4  =  i3  -  1  ;
label0:
 i2  =  i2  +  -1  ;
 $r1  = static_getter_java_lang_Integer_digits(gc_info, exception);
 $i5  =  i0  &  i4  ;
 $c6  = char__array_get(gc_info, $r1, $i5, exception);
if(*exception != 0) {
 
return 0; }
char__array_set(gc_info, r0, i2,  $c6 , exception);
if(*exception != 0) {
 
return 0; }
 i0  = ( i0  >>  i1  ) & 0x7fffffff;
if ( i0  !=  0   ) goto label0;
 $r2  =  -1 ;
 $i7  =  32  -  i2  ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r2, java_lang_String_initab850b60f96d11de8a390800200c9a660_a14_5_5_(gc_info,
  r0 ,
  i2 ,
  $i7 , exception));
if(*exception != 0) {
 
return 0; }
return  $r2 ;
  return 0;
}
__device__ double double__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*8);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( double *) &thisref_deref[offset]);
}
__device__ void double__array_set( char * gc_info, int thisref, int parameter0, double parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( double *) &thisref_deref[32+(parameter0*8)]) = parameter1;
}
__device__ int double__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 8)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2900);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  double__array_set(gc_info, thisref, i, 0, exception);
}
return thisref;
}
__device__ double java_lang_Math_abs8_8_( char * gc_info, double parameter0, int * exception){
double d0;
char $b0;
double $d1;
 d0  =  parameter0 ;
 $b0  = edu_syr_pcpratts_cmpg((double) d0 , (double) 0.0 );
if ( $b0  >  0   ) goto label0;
 $d1  =  0.0  -  d0  ;
goto label1;
label0:
 $d1  =  d0 ;
label1:
return  $d1 ;
  return 0;
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_assignCenters0_5_a12_( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int r0 = -1;
int i0;
int r1 = -1;
int i1;
int $r2 = -1;
int $i2;
int $r3 = -1;
int $r4 = -1;
int $r5 = -1;
int $r6 = -1;
int $i3;
int $r7 = -1;
int $r8 = -1;
double $d0;
double $d1;
double $d2;
int $r9 = -1;
int $i6;
int $i7;
int id;
char * mem;
char * trash;
char * mystery;
int count;
int old;
char * thisref_synch_deref;
if(thisref == -1){
  *exception = 21352;
  return;
}
id = getThreadId();
mem = edu_syr_pcpratts_gc_deref(gc_info, thisref);
trash = edu_syr_pcpratts_gc_deref(gc_info, 0) + 220;
mystery = trash - 8;
mem += 16;
count = 0;
while(count < 100){
  old = atomicCAS((int *) mem, -1 , id);
  *((int *) trash) = old;
  if(old == -1 || old == id){
  if ( thisref ==-1 ) {
 
    * exception = 11;
  }
  if ( * exception != 0 ) {
    edu_syr_pcpratts_exitMonitorMem ( gc_info , mem , old ) ;
    return;
  }
  thisref_synch_deref = edu_syr_pcpratts_gc_deref ( gc_info , thisref );
  * ( ( int * ) & thisref_synch_deref [ 20 ] ) = 20 ;
 r0  =  thisref ;
 i0  =  parameter0 ;
 r1  =  parameter1 ;
 $r2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $i2  = int__array_get(gc_info, $r2, i0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
if ( $i2  !=  -1   ) goto label0;
 $r3  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
double__array__array_set(gc_info, $r3, i0,  r1 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $r4  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
int__array_set(gc_info, $r4, i0,  0 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
goto label1;
label0:
 i1  =  0 ;
label3:
 $r5  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $r6  = double__array__array_get(gc_info, $r5, i0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $i3  = edu_syr_pcpratts_array_length(gc_info,  $r6 );
if ( i1  >=  $i3   ) goto label2;
 $r7  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_newCenters(gc_info, r0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $r8  = double__array__array_get(gc_info, $r7, i0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $d0  = double__array_get(gc_info, $r8, i1, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $d1  = double__array_get(gc_info, r1, i1, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $d2  =  $d0  +  $d1  ;
double__array_set(gc_info, $r8, i1,  $d2 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 i1  =  i1  +  1  ;
goto label3;
label2:
 $r9  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_m_summationCount(gc_info, r0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $i6  = int__array_get(gc_info, $r9, i0, exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
 $i7  =  $i6  +  1  ;
int__array_set(gc_info, $r9, i0,  $i7 , exception);
if(*exception != 0) {
 
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return ; }
label1:
edu_syr_pcpratts_exitMonitorMem(gc_info, mem, old);
return;
  }
 else {
    count++;
    if(count > 50 || (*((int *) mystery)) == 0){
      count = 0;
    }
  }
}
}
__device__ void java_lang_Exception_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_Throwable_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return;
}
__device__ char char__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( char *) &thisref_deref[offset]);
}
__device__ void char__array_set( char * gc_info, int thisref, int parameter0, char parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = 0;
*(( char *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int char__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 4335);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  char__array_set(gc_info, thisref, i, 0, exception);
}
return thisref;
}
__device__ int edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_initab850b60f96d11de8a390800200c9a660_13_13_( char * gc_info, int parameter0, int parameter1, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 2);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 3421);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, thisref, -1, exception);
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, thisref, -1, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
 r2  =  parameter1 ;
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_key(gc_info, r0,  r1 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_edu_syr_pcpratts_rootbeer_runtime_KeyValuePair_m_value(gc_info, r0,  r2 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception){
int r0 = -1;
int $r1 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2911);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, thisref, -1, exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 $r1  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new(gc_info,  8 , exception);
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0,  $r1 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0,  0 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_getLength5_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $i0;
 r0  =  thisref ;
 $i0  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
return  $i0 ;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 11550);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set(gc_info, thisref, i, -1, exception);
}
return thisref;
}
__device__ int double__array__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ void double__array__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int double__array__array_new_multi_array( char * gc_info, int dim0, int dim1, int * exception){
int total_size = (dim0 * 8) + 32;
int index0;
int aref0;
int index1;
int aref1;
int mod;
int thisref;
 char * thisref_deref;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2894);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, dim0);
for(index0 = 0; index0 < dim0; ++index0){
  aref0 = double__array_new(gc_info, dim1, exception);
  double__array__array_set(gc_info, thisref, index0, aref0, exception);
for(index1 = 0; index1 < dim1; ++index1){
  double__array_set(gc_info, aref0, index1, 0, exception);
}
}
return thisref;
}
__device__ void 
java_lang_System_arraycopy( char * gc_info, int src_handle, int srcPos, int dest_handle, int destPos, int length, int * exception){
  int i;
  int src_index;
  int dest_index;
   char * src_deref = edu_syr_pcpratts_gc_deref(gc_info, src_handle);
   char * dest_deref = edu_syr_pcpratts_gc_deref(gc_info, dest_handle);
  
  GC_OBJ_TYPE_TYPE src_type = edu_syr_pcpratts_gc_get_type(src_deref);
  GC_OBJ_TYPE_TYPE dest_type = edu_syr_pcpratts_gc_get_type(dest_deref);
  
  if(srcPos < destPos){
      if(0){
}
      else if(src_type == 4335 && dest_type == 4335){
        for(i = length - 1; i >= 0; --i){
          src_index = srcPos + i;
          dest_index = destPos + i;
        char__array_set(gc_info, dest_handle, dest_index, char__array_get(gc_info, src_handle, src_index, exception), exception);
        }
      }
  }
 else {
      if(0){
}
      else if(src_type == 4335 && dest_type == 4335){
        for(i = length - 1; i >= 0; --i){
          src_index = srcPos + i;
          dest_index = destPos + i;
        char__array_set(gc_info, dest_handle, dest_index, char__array_get(gc_info, src_handle, src_index, exception), exception);
        }
      }
  }
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_getLength5_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $i0;
 r0  =  thisref ;
 $i0  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
return  $i0 ;
  return 0;
}
__device__ int invoke_java_lang_StringBuilder_toString9_( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
GC_OBJ_TYPE_TYPE derived_type;
if(thisref == -1){
  *exception = -2;
return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
derived_type = edu_syr_pcpratts_gc_get_type(thisref_deref);
if(0){
}
else if(derived_type == 1){
return java_lang_Object_toString9_(gc_info, thisref, exception);
}
else if(derived_type == 16901){
return java_lang_StringBuilder_toString9_(gc_info, thisref, exception);
}
return -1;
}
__device__ int java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a660_9_( char * gc_info, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 32));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 21377);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 32);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_Throwable_cause(gc_info, thisref, -1, exception);
instance_setter_java_lang_Throwable_detailMessage(gc_info, thisref, -1, exception);
 r0  =  thisref ;
 r1  =  parameter0 ;
java_lang_RuntimeException_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  r1 , exception);
return r0;
  return 0;
}
__device__ int at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_get11_5_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int i0;
int $r1 = -1;
int $r2 = -1;
 r0  =  thisref ;
 i0  =  parameter0 ;
 $r1  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 $r2  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get(gc_info, $r1, i0, exception);
if(*exception != 0) {
 
return 0; }
return  $r2 ;
  return 0;
}
__device__ int java_lang_String_hashCode5_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int r1 = -1;
int i3;
int $i4;
int $i5;
char $c6;
 r0  =  thisref ;
 i0  = instance_getter_java_lang_String_hash(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 i1  = instance_getter_java_lang_String_count(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
if ( i0  !=  0   ) goto label0;
if ( i1  <=  0   ) goto label0;
 i2  = instance_getter_java_lang_String_offset(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 r1  = instance_getter_java_lang_String_value(gc_info, r0, exception);
if(*exception != 0) {
 
return 0; }
 i3  =  0 ;
label3:
if ( i3  >=  i1   ) goto label2;
 $i5  =  31  *  i0  ;
 $i4  =  i2 ;
 i2  =  i2  +  1  ;
 $c6  = char__array_get(gc_info, r1, $i4, exception);
if(*exception != 0) {
 
return 0; }
 i0  =  $i5  +  $c6  ;
 i3  =  i3  +  1  ;
goto label3;
label2:
instance_setter_java_lang_String_hash(gc_info, r0,  i0 , exception);
if(*exception != 0) {
 
return 0; }
label0:
return  i0 ;
  return 0;
}
__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int i0;
int i1;
int i2;
int $r0 = -1;
int r1 = -1;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 i2  =  parameter2 ;
 $r0  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r0, edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 r1  =  $r0 ;
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength(gc_info, r1,  i2 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex(gc_info, r1,  i0 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array(gc_info, r1,  i1 , exception);
if(*exception != 0) {
 
return 0; }
return  r1 ;
  return 0;
}
__device__ int java_lang_String_initab850b60f96d11de8a390800200c9a660_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int i0;
int i1;
int r1 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 64));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 1);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2905);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 64);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_String_value(gc_info, thisref, -1, exception);
instance_setter_java_lang_String_count(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_hash(gc_info, thisref, 0, exception);
instance_setter_java_lang_String_offset(gc_info, thisref, 0, exception);
 r0  =  thisref ;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 r1  =  parameter2 ;
instance_setter_java_lang_String_value(gc_info, r0,  r1 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_offset(gc_info, r0,  i0 , exception);
if(*exception != 0) {
 
return 0; }
instance_setter_java_lang_String_count(gc_info, r0,  i1 , exception);
if(*exception != 0) {
 
return 0; }
return r0;
  return 0;
}
__device__ double java_lang_Math_sqrt8_8_( char * gc_info, double parameter0, int * exception){
double d0;
double $d1;
 d0  =  parameter0 ;
 $d1  = java_lang_StrictMath_sqrt(gc_info,  d0 , exception);
if(*exception != 0) {
 
return 0; }
return  $d1 ;
  return 0;
}
__device__ int java_util_Arrays_copyOfRangea14_a14_5_5_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int r0 = -1;
int i0;
int i1;
int i2;
int $r1 = -1;
int r2 = -1;
int $r3 = -1;
int $r4 = -1;
int $r5 = -1;
int $r6 = -1;
int $r7 = -1;
int $i3;
int $i4;
int $i5;
 r0  =  parameter0 ;
 i0  =  parameter1 ;
 i1  =  parameter2 ;
 i2  =  i1  -  i0  ;
if ( i2  >=  0   ) goto label0;
 $r3  =  -1 ;
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 $r4  = java_lang_StringBuilder_append10_5_(gc_info,
 $r1,
  i0 , exception);
if(*exception != 0) {
 
return 0; }
 $r5  = java_lang_StringBuilder_append10_9_(gc_info,
 $r4,
  edu_syr_pcpratts_string_constant(gc_info, (char *) " > ", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $r6  = java_lang_StringBuilder_append10_5_(gc_info,
 $r5,
  i1 , exception);
if(*exception != 0) {
 
return 0; }
 $r7  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r6, exception);
if(*exception != 0) {
 
return 0; }
edu_syr_pcpratts_gc_assign (gc_info, 
&$r3, java_lang_IllegalArgumentException_initab850b60f96d11de8a390800200c9a660_9_(gc_info,
  $r7 , exception));
if(*exception != 0) {
 
return 0; }
 *exception =  $r3 ;
return 0;
label0:
 r2  = char__array_new(gc_info,  i2 , exception);
 $i3  = edu_syr_pcpratts_array_length(gc_info,  r0 );
 $i4  =  $i3  -  i0  ;
 $i5  = java_lang_Math_min5_5_5_(gc_info,  $i4 ,  i2 , exception);
if(*exception != 0) {
 
return 0; }
java_lang_System_arraycopy(gc_info,  r0 ,  i0 ,  r2 ,  0 ,  $i5 , exception);
if(*exception != 0) {
 
return 0; }
return  r2 ;
  return 0;
}
__device__ int java_lang_Object_toString9_( char * gc_info, int thisref, int * exception){
int r0 = -1;
int $r1 = -1;
int $r2 = -1;
int $r3 = -1;
int $r4 = -1;
int $r5 = -1;
int $i0;
int $r6 = -1;
int $r7 = -1;
int $r8 = -1;
 r0  =  thisref ;
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 $r2  = java_lang_Object_getClass(gc_info,
 r0, exception);
if(*exception != 0) {
 
return 0; }
 $r3  = java_lang_Class_getName(gc_info,
 $r2, exception);
if(*exception != 0) {
 
return 0; }
 $r4  = java_lang_StringBuilder_append10_9_(gc_info,
 $r1,
  $r3 , exception);
if(*exception != 0) {
 
return 0; }
 $r5  = java_lang_StringBuilder_append10_9_(gc_info,
 $r4,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "@", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $i0  = invoke_java_lang_Object_hashCode(gc_info,
 r0, exception);
if(*exception != 0) {
 
return 0; }
 $r6  = java_lang_Integer_toHexString9_5_(gc_info,  $i0 , exception);
if(*exception != 0) {
 
return 0; }
 $r7  = java_lang_StringBuilder_append10_9_(gc_info,
 $r5,
  $r6 , exception);
if(*exception != 0) {
 
return 0; }
 $r8  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r7, exception);
if(*exception != 0) {
 
return 0; }
return  $r8 ;
  return 0;
}
__device__ int invoke_java_lang_Object_hashCode( char * gc_info, int thisref, int * exception){
 char * thisref_deref;
GC_OBJ_TYPE_TYPE derived_type;
if(thisref == -1){
  *exception = -2;
return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
derived_type = edu_syr_pcpratts_gc_get_type(thisref_deref);
if(0){
}
else if(derived_type == 1){
return java_lang_Object_hashCode(gc_info, thisref, exception);
}
else if(derived_type == 2905){
return java_lang_String_hashCode5_(gc_info, thisref, exception);
}
return -1;
}
__device__ int edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_initab850b60f96d11de8a390800200c9a660_( char * gc_info, int * exception){
int r0 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 48));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 3608);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 48);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_array(gc_info, thisref, 0, exception);
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayIndex(gc_info, thisref, 0, exception);
instance_setter_edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_m_arrayLength(gc_info, thisref, 0, exception);
 r0  =  thisref ;
return r0;
  return 0;
}
__device__ int java_lang_Integer_stringSize5_5_( char * gc_info, int parameter0, int * exception){
int i0;
int i1;
int $r0 = -1;
int $i2;
int $i3;
 i0  =  parameter0 ;
 i1  =  0 ;
label1:
 $r0  = static_getter_java_lang_Integer_sizeTable(gc_info, exception);
 $i2  = int__array_get(gc_info, $r0, i1, exception);
if(*exception != 0) {
 
return 0; }
if ( i0  >  $i2   ) goto label0;
 $i3  =  i1  +  1  ;
return  $i3 ;
label0:
 i1  =  i1  +  1  ;
goto label1;
  return 0;
}
__device__ int java_lang_Integer_toHexString9_5_( char * gc_info, int parameter0, int * exception){
int i0;
int $r0 = -1;
 i0  =  parameter0 ;
 $r0  = java_lang_Integer_toUnsignedString9_5_5_(gc_info,  i0 ,  4 , exception);
if(*exception != 0) {
 
return 0; }
return  $r0 ;
  return 0;
}
__device__ int java_lang_Math_min5_5_5_( char * gc_info, int parameter0, int parameter1, int * exception){
int i0;
int i1;
int $i2;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
if ( i0  >  i1   ) goto label0;
 $i2  =  i0 ;
goto label1;
label0:
 $i2  =  i1 ;
label1:
return  $i2 ;
  return 0;
}
__device__ int java_lang_StringIndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a660_5_( char * gc_info, int parameter0, int * exception){
int r0 = -1;
int i0;
int $r1 = -1;
int $r2 = -1;
int $r3 = -1;
int $r4 = -1;
int thisref;
 char * thisref_deref;
thisref = -1;
edu_syr_pcpratts_gc_assign(gc_info, &thisref, edu_syr_pcpratts_gc_malloc(gc_info, 32));
if(thisref == -1){
  *exception = 21106;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 22978);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, 32);
edu_syr_pcpratts_gc_init_monitor(thisref_deref);
instance_setter_java_lang_Throwable_cause(gc_info, thisref, -1, exception);
instance_setter_java_lang_Throwable_detailMessage(gc_info, thisref, -1, exception);
 r0  =  thisref ;
 i0  =  parameter0 ;
 $r1  =  -1 ;
edu_syr_pcpratts_gc_assign (gc_info, 
&$r1, java_lang_StringBuilder_initab850b60f96d11de8a390800200c9a660_(gc_info, exception));
if(*exception != 0) {
 
return 0; }
 $r2  = java_lang_StringBuilder_append10_9_(gc_info,
 $r1,
  edu_syr_pcpratts_string_constant(gc_info, (char *) "String index out of range: ", exception) , exception);
if(*exception != 0) {
 
return 0; }
 $r3  = java_lang_StringBuilder_append10_5_(gc_info,
 $r2,
  i0 , exception);
if(*exception != 0) {
 
return 0; }
 $r4  = invoke_java_lang_StringBuilder_toString9_(gc_info,
 $r3, exception);
if(*exception != 0) {
 
return 0; }
java_lang_IndexOutOfBoundsException_initab850b60f96d11de8a390800200c9a66_body0_9_(gc_info,
 thisref,
  $r4 , exception);
return r0;
  return 0;
}
__device__ void at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_add0_11_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
int r2 = -1;
int i0;
int $r3 = -1;
int $i1;
int $i2;
int $i3;
int $i4;
int $r5 = -1;
int $i5;
int $i6;
int $i7;
int $i8;
int $r6 = -1;
int $r7 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
 $r3  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i1  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set(gc_info, $r3, $i1,  r1 , exception);
if(*exception != 0) {
 
return ; }
 $i2  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i3  =  $i2  +  1  ;
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0,  $i3 , exception);
if(*exception != 0) {
 
return ; }
 $i4  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r5  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i5  = edu_syr_pcpratts_array_length(gc_info,  $r5 );
if ( $i4  !=  $i5   ) goto label0;
 $i6  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $i7  =  $i6  *  2  ;
 r2  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_new(gc_info,  $i7 , exception);
 i0  =  0 ;
label2:
 $i8  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_index(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
if ( i0  >=  $i8   ) goto label1;
 $r6  = instance_getter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0, exception);
if(*exception != 0) {
 
return ; }
 $r7  = at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_get(gc_info, $r6, i0, exception);
if(*exception != 0) {
 
return ; }
at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVector__array_set(gc_info, r2, i0,  $r7 , exception);
if(*exception != 0) {
 
return ; }
 i0  =  i0  +  1  ;
goto label2;
label1:
instance_setter_at_illecker_hama_hybrid_examples_kmeans_DenseDoubleVectorList_m_values(gc_info, r0,  r2 , exception);
if(*exception != 0) {
 
return ; }
label0:
return;
}
__device__ void java_lang_Throwable_initab850b60f96d11de8a390800200c9a66_body0_9_( char * gc_info, int thisref, int parameter0, int * exception){
int r0 = -1;
int r1 = -1;
 r0  =  thisref ;
 r1  =  parameter0 ;
instance_setter_java_lang_Throwable_cause(gc_info, r0,  r0 , exception);
if(*exception != 0) {
 
return ; }
java_lang_Throwable_fillInStackTrace(gc_info,
 r0, exception);
if(*exception != 0) {
 
return ; }
instance_setter_java_lang_Throwable_detailMessage(gc_info, r0,  r1 , exception);
if(*exception != 0) {
 
return ; }
return;
}
__device__ void java_lang_Integer_getChars0_5_5_a14_( char * gc_info, int parameter0, int parameter1, int parameter2, int * exception){
int i0;
int i1;
int r0 = -1;
int i2;
char c3;
int i4;
int i5;
int $i6;
int $i7;
int $i8;
int $i9;
int $i10;
int i11;
int $r1 = -1;
char $c12;
int $r2 = -1;
char $c13;
int $i14;
int i15;
int $i16;
int $i17;
int $i18;
int i19;
int $r3 = -1;
char $c20;
int i21;
 i0  =  parameter0 ;
 i1  =  parameter1 ;
 r0  =  parameter2 ;
 i2  =  i1 ;
 c3  =  0 ;
if ( i0  >=  0   ) goto label0;
 c3  =  45 ;
 i0  = - i0 ;
label0:
if ( i0  <  65536   ) goto label1;
 i4  =  i0  /  100  ;
 $i6  =  i4  <<  6  ;
 $i7  =  i4  <<  5  ;
 $i8  =  $i6  +  $i7  ;
 $i9  =  i4  <<  2  ;
 $i10  =  $i8  +  $i9  ;
 i5  =  i0  -  $i10  ;
 i0  =  i4 ;
 i11  =  i2  +  -1  ;
 $r1  = static_getter_java_lang_Integer_DigitOnes(gc_info, exception);
 $c12  = char__array_get(gc_info, $r1, i5, exception);
if(*exception != 0) {
 
return ; }
char__array_set(gc_info, r0, i11,  $c12 , exception);
if(*exception != 0) {
 
return ; }
 i2  =  i11  +  -1  ;
 $r2  = static_getter_java_lang_Integer_DigitTens(gc_info, exception);
 $c13  = char__array_get(gc_info, $r2, i5, exception);
if(*exception != 0) {
 
return ; }
char__array_set(gc_info, r0, i2,  $c13 , exception);
if(*exception != 0) {
 
return ; }
goto label0;
label1:
 $i14  =  i0  *  52429  ;
 i15  = ( $i14  >>  19  ) & 0x7fffffff;
 $i16  =  i15  <<  3  ;
 $i17  =  i15  <<  1  ;
 $i18  =  $i16  +  $i17  ;
 i19  =  i0  -  $i18  ;
 i2  =  i2  +  -1  ;
 $r3  = static_getter_java_lang_Integer_digits(gc_info, exception);
 $c20  = char__array_get(gc_info, $r3, i19, exception);
if(*exception != 0) {
 
return ; }
char__array_set(gc_info, r0, i2,  $c20 , exception);
if(*exception != 0) {
 
return ; }
 i0  =  i15 ;
if ( i0  !=  0   ) goto label1;
goto label4;
label4:
if ( c3  ==  0   ) goto label5;
 i21  =  i2  +  -1  ;
char__array_set(gc_info, r0, i21,  c3 , exception);
if(*exception != 0) {
 
return ; }
label5:
return;
}
__device__ int java_lang_String__array_get( char * gc_info, int thisref, int parameter0, int * exception){
int offset;
int length;
 char * thisref_deref;
offset = 32+(parameter0*4);
if(thisref == -1){
  *exception = 21352;
  return 0;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return 0;
}
return *(( int *) &thisref_deref[offset]);
}
__device__ void java_lang_String__array_set( char * gc_info, int thisref, int parameter0, int parameter1, int * exception){
int length;
 char * thisref_deref;
  if(thisref == -1){
    *exception = 21352;
    return;
  }
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
length = edu_syr_pcpratts_getint(thisref_deref, 12);
if(parameter0 < 0 || parameter0 >= length){
  *exception = edu_syr_pcpratts_rootbeer_runtimegpu_GpuException_arrayOutOfBounds(gc_info, parameter0, thisref, length, exception);  return;
}
*(( int *) &thisref_deref[32+(parameter0*4)]) = parameter1;
}
__device__ int java_lang_String__array_new( char * gc_info, int size, int * exception){
int i;
int total_size;
int mod;
int thisref;
 char * thisref_deref;
total_size = (size * 4)+ 32;
mod = total_size % 8;
if(mod != 0)
  total_size += (8 - mod);
thisref = edu_syr_pcpratts_gc_malloc(gc_info, total_size);
if(thisref == -1){
  *exception = 21352;
  return -1;
}
thisref_deref = edu_syr_pcpratts_gc_deref(gc_info, thisref);
edu_syr_pcpratts_gc_set_count(thisref_deref, 0);
edu_syr_pcpratts_gc_set_color(thisref_deref, COLOR_GREY);
edu_syr_pcpratts_gc_set_type(thisref_deref, 2903);
edu_syr_pcpratts_gc_set_ctor_used(thisref_deref, 1);
edu_syr_pcpratts_gc_set_size(thisref_deref, total_size);
edu_syr_pcpratts_setint(thisref_deref, 12, size);
for(i = 0; i < size; ++i){
  java_lang_String__array_set(gc_info, thisref, i, -1, exception);
}
return thisref;
}
__device__ int
edu_syr_pcpratts_classConstant(int type_num){
  int * temp = (int *) m_Local[2];   
  return temp[type_num];
}
__device__  char *
edu_syr_pcpratts_gc_deref(char * gc_info, int handle){
  char * data_arr = (char * ) m_Local[0];
  long long lhandle = handle;
  lhandle = lhandle << 4;
  return &data_arr[lhandle];
}
__device__ int
edu_syr_pcpratts_gc_malloc(char * gc_info, int size){
  unsigned long long space_size = m_Local[1];
  unsigned long long ret = edu_syr_pcpratts_gc_malloc_no_fail(gc_info, size);
  unsigned long long end = ret + size + 8L;
  if(end >= space_size){
    return -1;
  }
  return (int) (ret >> 4);
}
__device__ unsigned long long
edu_syr_pcpratts_gc_malloc_no_fail(char * gc_info, int size){
  unsigned long long * addr = (unsigned long long *) (gc_info + TO_SPACE_FREE_POINTER_OFFSET);
  if(size % 16 != 0){
    size += (16 - (size %16));
  }
  unsigned long long ret;
  ret = atomicAdd(addr, size);
  return ret;
}
__device__  void
edu_syr_pcpratts_gc_init(char * to_space, size_t space_size, int * java_lang_class_refs){
  if(threadIdx.x == 0){
    m_Local[0] = (size_t) to_space;
    m_Local[1] = (size_t) space_size;
    m_Local[2] = (size_t) java_lang_class_refs;
  }
}
__global__ void entry(char * gc_info, char * to_space, int * handles, 
  long long * to_space_free_ptr, long long * space_size, int * exceptions,
  int * java_lang_class_refs, HostDeviceInterface * h_d_interface,
  int num_blocks) {
  
  host_device_interface = h_d_interface;
  
  
  edu_syr_pcpratts_gc_init(to_space, *space_size, java_lang_class_refs);
  __syncthreads();
  int loop_control = blockIdx.x * blockDim.x + threadIdx.x;
  if(loop_control >= num_blocks){
  
    return;
  }
 else {
    int handle = handles[loop_control];
    int exception = 0;   
    at_illecker_hama_hybrid_examples_kmeans_KMeansHybridKernelOld_gpuMethod0_(gc_info, handle, &exception);
    exceptions[loop_control] = exception;
  
    unsigned long long * addr = ( unsigned long long * ) (gc_info + TO_SPACE_FREE_POINTER_OFFSET);
    *to_space_free_ptr = *addr;
  }
}

